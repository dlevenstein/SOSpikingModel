#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * CAdLIFfunction_STDP.cu
 *
 * Code generation for function 'CAdLIFfunction_STDP'
 *
 */

/* Include files */
#include "MWCudaDimUtility.h"
#include <stdlib.h>
#include <math.h>
#include "rt_nonfinite.h"
#include "CAdLIFfunction_STDP.h"
#include "CAdLIFfunction_STDP_emxutil.h"
#include "CAdLIFfunction_STDP_data.h"

/* Variable Definitions */
static emlrtRTEInfo emlrtRTEI = { 403, /* lineNo */
  9,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo emlrtMCI = { 537,   /* lineNo */
  1,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo b_emlrtMCI = { 538, /* lineNo */
  5,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo c_emlrtMCI = { 539, /* lineNo */
  5,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo d_emlrtMCI = { 540, /* lineNo */
  5,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo e_emlrtMCI = { 541, /* lineNo */
  5,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo f_emlrtMCI = { 541, /* lineNo */
  25,                                  /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo g_emlrtMCI = { 541, /* lineNo */
  45,                                  /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo h_emlrtMCI = { 542, /* lineNo */
  5,                                   /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

static emlrtMCInfo i_emlrtMCI = { 542, /* lineNo */
  32,                                  /* colNo */
  "CAdLIFfunction_STDP",               /* fName */
  "C:\\Users\\jgorn\\OneDrive\\Documents\\GitHub\\SOSpikingModel\\AnalysisScripts\\CAdLIFfunction_STDP.m"/* pName */
};

/* Function Declarations */
static __global__ void CAdLIFfunction_STDP_kernel1(real_T *recordIntervals,
  real_T *saveINT);
static void b_plot(const mxArray *b, const mxArray *c, const mxArray *d,
                   emlrtMCInfo *location);
static void checkCudaError(hipError_t errCode, const char * file, uint32_T line);
static void figure(emlrtMCInfo *location);
static void hold(const mxArray *b, emlrtMCInfo *location);
static void plot(const mxArray *b, const mxArray *c, const mxArray *d, const
                 mxArray *e, const mxArray *f, emlrtMCInfo *location);
static real_T rt_roundd_snf(real_T u);
static void title(const mxArray *b, emlrtMCInfo *location);
static void xlabel(const mxArray *b, emlrtMCInfo *location);
static void xlim(const mxArray *b, emlrtMCInfo *location);
static void ylabel(const mxArray *b, emlrtMCInfo *location);
static void ylim(const mxArray *b, emlrtMCInfo *location);

/* Function Definitions */
static __global__ __launch_bounds__(32, 1) void CAdLIFfunction_STDP_kernel1
  (real_T *recordIntervals, real_T *saveINT)
{
  uint32_T threadId;
  int32_T ii;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  ii = (int32_T)threadId;
  if (!(ii >= 2)) {
    saveINT[ii << 1] = recordIntervals[ii << 1];
    saveINT[1 + (ii << 1)] = recordIntervals[1 + (ii << 1)];
  }
}

static void b_plot(const mxArray *b, const mxArray *c, const mxArray *d,
                   emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  pArrays[0] = b;
  pArrays[1] = c;
  pArrays[2] = d;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 3, pArrays, "plot", true,
                        location);
}

static void checkCudaError(hipError_t errCode, const char * file, uint32_T line)
{
  emlrtRTEInfo rtInfo;
  char_T *pn;
  char_T *fn;
  char_T *brk;
  uint32_T len;
  if (errCode != hipSuccess) {
    len = strlen(file);
    pn = (char_T *)calloc(len + 1U, 1U);
    fn = (char_T *)calloc(len + 1U, 1U);
    memcpy(pn, file, len);
    memcpy(fn, file, len);
    brk = strrchr(fn, '.');
    *brk = '\x00';
    brk = strrchr(fn, '/');
    if (brk == NULL) {
      brk = strrchr(fn, '\\');
    }

    if (brk == NULL) {
      brk = fn;
    } else {
      brk++;
    }

    rtInfo.lineNo = (int32_T)line;
    rtInfo.colNo = 0;
    rtInfo.fName = brk;
    rtInfo.pName = pn;
    emlrtCUDAError(errCode, hipGetErrorName(errCode), hipGetErrorString
                   (errCode), &rtInfo, emlrtRootTLSGlobal);
  }
}

static void figure(emlrtMCInfo *location)
{
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 0, NULL, "figure", true,
                        location);
}

static void hold(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "hold", true,
                        location);
}

static void plot(const mxArray *b, const mxArray *c, const mxArray *d, const
                 mxArray *e, const mxArray *f, emlrtMCInfo *location)
{
  const mxArray *pArrays[5];
  pArrays[0] = b;
  pArrays[1] = c;
  pArrays[2] = d;
  pArrays[3] = e;
  pArrays[4] = f;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 5, pArrays, "plot", true,
                        location);
}

static real_T rt_roundd_snf(real_T u)
{
  real_T y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

static void title(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "title", true,
                        location);
}

static void xlabel(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "xlabel", true,
                        location);
}

static void xlim(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "xlim", true,
                        location);
}

static void ylabel(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "ylabel", true,
                        location);
}

static void ylim(const mxArray *b, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = b;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, NULL, 1, &pArray, "ylim", true,
                        location);
}

void CAdLIFfunction_STDP(const struct0_T *PopParams, const struct1_T *TimeParams,
  struct2_T *SimValues)
{
  real_T recordIntervals[4];
  emxArray_real_T *tSAVE;
  int32_T i0;
  emxArray_int8_T *recordVALs;
  real_T ndbl;
  real_T apnd;
  real_T cdiff;
  int32_T loopUpperBound;
  real_T saveINT[4];
  int32_T n;
  int32_T ii;
  int32_T loop_ub;
  real_T EPopNum;
  int32_T nm1d2;
  int32_T k;
  real_T PopNum;
  real_T dt;
  emxArray_real_T *r0;
  int32_T varargin_2;
  emxArray_real_T *r1;
  emxArray_real_T *r2;
  emxArray_real_T *EE_mat;
  emxArray_real_T *II_mat;
  emxArray_real_T *IE_mat;
  emxArray_real_T *EI_mat;
  emxArray_real_T *Ecells;
  emxArray_real_T *y;
  emxArray_boolean_T *EcellIDX;
  emxArray_real_T *Icells;
  emxArray_real_T *b_y;
  real_T absa;
  emxArray_boolean_T *IcellIDX;
  emxArray_int32_T *r3;
  emxArray_int32_T *r4;
  emxArray_real_T *r;
  uint32_T uv0[2];
  emxArray_boolean_T *d;
  emxArray_boolean_T *r5;
  int32_T iv0[2];
  int32_T b_loopUpperBound;
  int32_T mA;
  emxArray_int32_T *r6;
  emxArray_int32_T *r7;
  emxArray_int32_T *r8;
  emxArray_real_T *b_r;
  emxArray_boolean_T *b_d;
  emxArray_boolean_T *r9;
  emxArray_int32_T *r10;
  emxArray_int32_T *r11;
  emxArray_int32_T *r12;
  emxArray_real_T *c_r;
  emxArray_int32_T *r13;
  emxArray_int32_T *r14;
  emxArray_real_T *d_r;
  emxArray_boolean_T *isconnected;
  emxArray_real_T *E_L;
  emxArray_real_T *g_L;
  emxArray_real_T *C;
  emxArray_real_T *I_e;
  emxArray_real_T *V_th;
  emxArray_real_T *V_reset;
  emxArray_real_T *t_ref;
  emxArray_real_T *sigma;
  emxArray_real_T *theta;
  emxArray_real_T *E_w;
  emxArray_real_T *b_w;
  emxArray_real_T *delta_T;
  emxArray_real_T *gwnorm;
  emxArray_real_T *w_r;
  emxArray_real_T *b;
  emxArray_real_T *E_e;
  emxArray_real_T *E_i;
  emxArray_real_T *tau_s;
  emxArray_real_T *ELearningRate;
  emxArray_real_T *ILearningRate;
  emxArray_real_T *tauSTDP;
  emxArray_real_T *alpha;
  emxArray_real_T *V;
  emxArray_real_T *g_e;
  emxArray_real_T *g_i;
  emxArray_real_T *g_w;
  emxArray_real_T *a_w;
  emxArray_real_T *s;
  emxArray_real_T *w;
  emxArray_real_T *X_t;
  emxArray_real_T *t_r;
  emxArray_real_T *x;
  emxArray_boolean_T *b_x;
  emxArray_int32_T *b_ii;
  int32_T nx;
  int32_T idx;
  emxArray_real32_T *spikes;
  emxArray_int32_T *r15;
  emxArray_int32_T *r16;
  emxArray_real_T *c_x;
  emxArray_real_T *r17;
  emxArray_real_T *dX;
  boolean_T p;
  emxArray_real_T *e_r;
  uint32_T savecounter;
  uint32_T weightcounter;
  emxArray_real_T *PreIspikes;
  emxArray_real_T *PostEspikes;
  emxArray_real_T *r18;
  emxArray_real_T *r19;
  emxArray_real_T *r20;
  emxArray_int32_T *r21;
  emxArray_int32_T *r22;
  emxArray_int32_T *r23;
  emxArray_int32_T *r24;
  emxArray_int32_T *r25;
  emxArray_int32_T *r26;
  emxArray_int32_T *r27;
  emxArray_boolean_T *d_x;
  emxArray_real_T *e_x;
  emxArray_real_T *f_x;
  emxArray_real_T *g_x;
  emxArray_boolean_T *h_x;
  emxArray_boolean_T *i_x;
  emxArray_int32_T *c_ii;
  cell_wrap_1 reshapes[2];
  emxArray_boolean_T *j_x;
  emxArray_real_T *b_C;
  emxArray_real_T *c_C;
  emxArray_real_T *d_C;
  emxArray_real_T *e_C;
  emxArray_real_T *b_s;
  emxArray_real_T *k_x;
  emxArray_real_T *b_EI_mat;
  emxArray_real_T *c_EI_mat;
  emxArray_real_T *b_t_r;
  int32_T l_x;
  int32_T tt;
  real_T f_r;
  real_T g_r;
  const mxArray *c_y;
  const mxArray *m0;
  emxArray_int32_T *b_idx;
  boolean_T b0;
  int32_T br;
  emxArray_boolean_T *b_b;
  emxArray_real32_T *b_spikes;
  int32_T nA;
  emxArray_real32_T *u;
  emxArray_boolean_T *r28;
  real32_T *pData;
  emxArray_real32_T *b_u;
  const mxArray *d_y;
  const mxArray *e_y;
  static const int32_T iv1[2] = { 1, 2 };

  static const char_T c_u[2] = { 'k', '.' };

  const mxArray *f_y;
  static const int32_T iv2[2] = { 1, 10 };

  static const char_T d_u[10] = { 'M', 'a', 'r', 'k', 'e', 'r', 's', 'i', 'z',
    'e' };

  const mxArray *g_y;
  static const int32_T iv3[2] = { 1, 2 };

  static const char_T e_u[2] = { 'o', 'n' };

  static const int32_T iv4[2] = { 1, 2 };

  real_T *b_pData;
  real_T V0range[2];
  static const int32_T iv5[2] = { 1, 2 };

  static const int32_T iv6[2] = { 1, 9 };

  static const char_T f_u[9] = { 'T', 'i', 'm', 'e', ' ', '(', 'm', 's', ')' };

  static const int32_T iv7[2] = { 1, 9 };

  static const char_T g_u[9] = { 'N', 'e', 'u', 'r', 'o', 'n', ' ', 'I', 'D' };

  static const int32_T iv8[2] = { 1, 11 };

  static const char_T h_u[11] = { 'R', 'a', 's', 't', 'e', 'r', ' ', 'P', 'l',
    'o', 't' };

  static const int32_T iv9[2] = { 1, 2 };

  static const int32_T iv10[2] = { 1, 2 };

  emxArray_boolean_T *c_idx;
  emxArray_real32_T *c_spikes;
  real_T *gpu_recordIntervals;
  real_T *gpu_saveINT;
  boolean_T saveINT_dirtyOnGpu;
  boolean_T exitg1;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  CUDACHECK(hipMalloc(&gpu_saveINT, 32ULL));
  CUDACHECK(hipMalloc(&gpu_recordIntervals, 32ULL));
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);

  /* Conductance-Based Adapting LIF Model, Euler Mayurama   implementation  */
  /* with Conductance-based Jump-decay STDP synapses */
  /* by Jonathan Gornet and DLevenstein */
  /* Last update: 4/8/2018 */
  /* INPUTS */
  /*    PopParams       a structure that gives all the parameters of the population */
  /*        .EPopNum	Number of excitatory neurons */
  /*        .IPopNum	Number of inhibitory neurons */
  /*        .E_L     	Reversal potential of the leak current (i.e. Vrest) */
  /*        .g_L     	Conductance of the leak current  */
  /*        .C          Membrane Capacitance */
  /*        .I_e        Input current to the population. Can either be a */
  /*                    constant, input to [E I] populations, */
  /*                    or a function I_e(t) that returns input at time t */
  /*                    time t. Add:  */
  /*        .V_th       Membrane Threshold */
  /*        .V_reset    Reset Potential */
  /*  */
  /*        .sigma      Standard deviation of noise */
  /*        .theta      TimeScale of noise */
  /*   */
  /*        .E_e        Reversal potential of excitatory synapses */
  /*        .E_i        Reversal potential of inhibitory synapses */
  /*        .tau_s      Time of synaptic decay */
  /*   */
  /*        .tau_a      Adaptation time constant */
  /*        .t_ref      Refractory period */
  /*        .delta_T    Adaptation threshold softness */
  /*        .gwnorm     Adaptation normalizer */
  /*        .b          Adaptation activation strength */
  /*        .bw         Time of adaptative decay */
  /*        .w_r        Adaptation Rest */
  /*  */
  /*                    SYNAPTIC WEIGHT (conductance jump: nS?) */
  /*        .Wee        E->E synapse weight */
  /*        .Wii        I->I synapse weight */
  /*        .Wie        E->I synapse weight */
  /*        .Wei        I->E synapse weight */
  /*                    CONNECTION PROBABILITY (Expected In-Degree) */
  /*        .Kee        E->E */
  /*        .Kii        I->I */
  /*        .Kie        E->I */
  /*        .Kei        I->E */
  /*  */
  /*        .p0spike    (optional) proportion of neurons spiking at t0 (default:0) */
  /*    TimeParams */
  /*        .dt        timestep (ms) */
  /*        .SimTime   total simulation time (ms) */
  /*    'showfig'       (optional) show the figure? (default:true) */
  /*    'showprogress'  (optional) show time counter of progress (default:false) */
  /*    'onsettime'     (optional) duration of (removed?) onset time (default: 0ms) */
  /*    'save_dt'       (optional) dt for the saved output (default: 0.1ms) */
  /*  */
  /* -------------------------------------------------------------------------- */
  /* -------------------------------------------------------------------------- */
  /* Parse optional inputs */
  /*  p = inputParser; */
  /*  addParameter(p,'showfig',true,@islogical) */
  /*  addParameter(p,'showprogress',false,@islogical) */
  /*  addParameter(p,'onsettime',0,@isnumeric) */
  /*  addParameter(p,'save_dt',0.5,@isnumeric) */
  /*  addParameter(p,'save_weights',10,@isnumeric) */
  /*  addParameter(p,'cellout',false,@islogical) */
  /*  addParameter(p,'recordInterval',[],@isnumeric) */
  /*  parse(p,varargin{:}) */
  /*  SHOWFIG = p.Results.showfig; */
  /*  SHOWPROGRESS = p.Results.showprogress; */
  /*  onsettime = p.Results.onsettime; */
  /*  save_dt = p.Results.save_dt; */
  /*  save_weights = p.Results.save_weights; */
  /*  cellout = p.Results.cellout; */
  /*  recordIntervals = p.Results.recordInterval; */
  recordIntervals[0] = 0.0;
  recordIntervals[2] = TimeParams->SimTime - 1000.0;
  recordIntervals[1] = 1000.0;
  recordIntervals[3] = TimeParams->SimTime;
  emxInit_real_T(&tSAVE, 2, true);
  if (rtIsNaN(TimeParams->dt) || rtIsNaN(TimeParams->SimTime)) {
    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = 1;
    emxEnsureCapacity_real_T(tSAVE, i0);
    tSAVE->data[0] = rtNaN;
  } else if ((TimeParams->dt == 0.0) || ((0.0 < TimeParams->SimTime) &&
              (TimeParams->dt < 0.0)) || ((TimeParams->SimTime < 0.0) &&
              (TimeParams->dt > 0.0))) {
    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = 0;
    emxEnsureCapacity_real_T(tSAVE, i0);
  } else if (rtIsInf(TimeParams->SimTime) && (rtIsInf(TimeParams->dt) || (0.0 ==
    TimeParams->SimTime))) {
    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = 1;
    emxEnsureCapacity_real_T(tSAVE, i0);
    tSAVE->data[0] = rtNaN;
  } else if (rtIsInf(TimeParams->dt)) {
    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = 1;
    emxEnsureCapacity_real_T(tSAVE, i0);
    tSAVE->data[0] = 0.0;
  } else if (floor(TimeParams->dt) == TimeParams->dt) {
    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = (int32_T)floor(TimeParams->SimTime / TimeParams->dt) + 1;
    emxEnsureCapacity_real_T(tSAVE, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(TimeParams->SimTime /
          TimeParams->dt); loopUpperBound++) {
      tSAVE->data[tSAVE->size[0] * loopUpperBound] = TimeParams->dt * (real_T)
        loopUpperBound;
    }
  } else {
    ndbl = floor(TimeParams->SimTime / TimeParams->dt + 0.5);
    apnd = ndbl * TimeParams->dt;
    if (TimeParams->dt > 0.0) {
      cdiff = apnd - TimeParams->SimTime;
    } else {
      cdiff = TimeParams->SimTime - apnd;
    }

    if (fabs(cdiff) < 4.4408920985006262E-16 * fabs(TimeParams->SimTime)) {
      ndbl++;
      apnd = TimeParams->SimTime;
    } else if (cdiff > 0.0) {
      apnd = (ndbl - 1.0) * TimeParams->dt;
    } else {
      ndbl++;
    }

    if (ndbl >= 0.0) {
      n = (int32_T)ndbl;
    } else {
      n = 0;
    }

    i0 = tSAVE->size[0] * tSAVE->size[1];
    tSAVE->size[0] = 1;
    tSAVE->size[1] = n;
    emxEnsureCapacity_real_T(tSAVE, i0);
    if (n > 0) {
      tSAVE->data[0] = 0.0;
      if (n > 1) {
        tSAVE->data[n - 1] = apnd;
        nm1d2 = (n - 1) / 2;
        for (k = 0; k <= nm1d2 - 2; k++) {
          ndbl = (1.0 + (real_T)k) * TimeParams->dt;
          tSAVE->data[1 + k] = ndbl;
          tSAVE->data[(n - k) - 2] = apnd - ndbl;
        }

        if (nm1d2 << 1 == n - 1) {
          tSAVE->data[nm1d2] = apnd / 2.0;
        } else {
          ndbl = (real_T)nm1d2 * TimeParams->dt;
          tSAVE->data[nm1d2] = ndbl;
          tSAVE->data[nm1d2 + 1] = apnd - ndbl;
        }
      }
    }
  }

  emxInit_int8_T(&recordVALs, 2, true);
  i0 = recordVALs->size[0] * recordVALs->size[1];
  recordVALs->size[0] = 1;
  recordVALs->size[1] = tSAVE->size[1];
  emxEnsureCapacity_int8_T(recordVALs, i0);
  for (loopUpperBound = 0; loopUpperBound < tSAVE->size[1]; loopUpperBound++) {
    recordVALs->data[loopUpperBound] = 0;
  }

  CUDACHECK(hipMemcpy((void *)gpu_recordIntervals, (void *)&recordIntervals[0],
                       32ULL, hipMemcpyHostToDevice));
  CAdLIFfunction_STDP_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (gpu_recordIntervals, gpu_saveINT);
  CUDACHECK(hipGetLastError());
  saveINT_dirtyOnGpu = true;
  for (ii = 0; ii < 2; ii++) {
    loopUpperBound = tSAVE->size[1];
    for (loop_ub = 0; loop_ub <= loopUpperBound - 2; loop_ub++) {
      if (saveINT_dirtyOnGpu) {
        CUDACHECK(hipMemcpy((void *)&saveINT[0], (void *)gpu_saveINT, 32ULL,
                             hipMemcpyDeviceToHost));
        saveINT_dirtyOnGpu = false;
      }

      if ((tSAVE->data[loop_ub] >= saveINT[ii << 1]) && (tSAVE->data[loop_ub] <=
           saveINT[1 + (ii << 1)])) {
        recordVALs->data[loop_ub] = 1;
      }
    }
  }

  emxFree_real_T(&tSAVE);

  /*  */
  /* -------------------------------------------------------------------------- */
  /* Simulation Parameters */
  EPopNum = PopParams->EPopNum;

  /* Number of excitatory neurons */
  /* Number of excitatory neurons */
  PopNum = PopParams->EPopNum + PopParams->IPopNum;

  /* Number of all neurons */
  /* Simulation Time (ms) */
  dt = TimeParams->dt;

  /* differential (ms) */
  /* Calculate time vector from time parameters */
  emxInit_real_T(&r0, 2, true);
  if (rtIsNaN(TimeParams->dt) || rtIsNaN(TimeParams->SimTime)) {
    i0 = r0->size[0] * r0->size[1];
    r0->size[1] = 1;
    emxEnsureCapacity_real_T(r0, i0);
  } else if ((TimeParams->dt == 0.0) || ((-0.0 < TimeParams->SimTime) &&
              (TimeParams->dt < 0.0)) || ((TimeParams->SimTime < -0.0) &&
              (TimeParams->dt > 0.0))) {
    i0 = r0->size[0] * r0->size[1];
    r0->size[1] = 0;
    emxEnsureCapacity_real_T(r0, i0);
  } else if (rtIsInf(TimeParams->SimTime) && (rtIsInf(TimeParams->dt) || (-0.0 ==
    TimeParams->SimTime))) {
    i0 = r0->size[0] * r0->size[1];
    r0->size[1] = 1;
    emxEnsureCapacity_real_T(r0, i0);
  } else if (rtIsInf(TimeParams->dt)) {
    i0 = r0->size[0] * r0->size[1];
    r0->size[1] = 1;
    emxEnsureCapacity_real_T(r0, i0);
  } else if (floor(TimeParams->dt) == TimeParams->dt) {
    i0 = r0->size[0] * r0->size[1];
    r0->size[0] = 1;
    r0->size[1] = (int32_T)floor(TimeParams->SimTime / TimeParams->dt) + 1;
    emxEnsureCapacity_real_T(r0, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(TimeParams->SimTime /
          TimeParams->dt); loopUpperBound++) {
      r0->data[r0->size[0] * loopUpperBound] = TimeParams->dt * (real_T)
        loopUpperBound;
    }
  } else {
    ndbl = floor(TimeParams->SimTime / TimeParams->dt + 0.5);
    apnd = ndbl * TimeParams->dt;
    if (TimeParams->dt > 0.0) {
      cdiff = apnd - TimeParams->SimTime;
    } else {
      cdiff = TimeParams->SimTime - apnd;
    }

    if (fabs(cdiff) < 4.4408920985006262E-16 * fabs(TimeParams->SimTime)) {
      ndbl++;
      apnd = TimeParams->SimTime;
    } else if (cdiff > 0.0) {
      apnd = (ndbl - 1.0) * TimeParams->dt;
    } else {
      ndbl++;
    }

    if (ndbl >= 0.0) {
      n = (int32_T)ndbl;
    } else {
      n = 0;
    }

    i0 = r0->size[0] * r0->size[1];
    r0->size[0] = 1;
    r0->size[1] = n;
    emxEnsureCapacity_real_T(r0, i0);
    if (n > 0) {
      r0->data[0] = -0.0;
      if (n > 1) {
        r0->data[n - 1] = apnd;
        nm1d2 = (n - 1) / 2;
        for (k = 0; k <= nm1d2 - 2; k++) {
          ndbl = (1.0 + (real_T)k) * dt;
          r0->data[1 + k] = ndbl;
          r0->data[(n - k) - 2] = apnd - ndbl;
        }

        if (nm1d2 << 1 == n - 1) {
          r0->data[nm1d2] = apnd / 2.0;
        } else {
          ndbl = (real_T)nm1d2 * TimeParams->dt;
          r0->data[nm1d2] = ndbl;
          r0->data[nm1d2 + 1] = apnd - ndbl;
        }
      }
    }
  }

  varargin_2 = r0->size[1];

  /* Time Steps (simulated) */
  emxInit_real_T(&r1, 2, true);
  if (rtIsNaN(TimeParams->SimTime)) {
    i0 = r1->size[0] * r1->size[1];
    r1->size[1] = 1;
    emxEnsureCapacity_real_T(r1, i0);
  } else if (TimeParams->SimTime < 0.0) {
    i0 = r1->size[0] * r1->size[1];
    r1->size[1] = 0;
    emxEnsureCapacity_real_T(r1, i0);
  } else if (rtIsInf(TimeParams->SimTime) && (0.0 == TimeParams->SimTime)) {
    i0 = r1->size[0] * r1->size[1];
    r1->size[1] = 1;
    emxEnsureCapacity_real_T(r1, i0);
  } else {
    i0 = r1->size[0] * r1->size[1];
    r1->size[0] = 1;
    r1->size[1] = (int32_T)floor(TimeParams->SimTime) + 1;
    emxEnsureCapacity_real_T(r1, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)TimeParams->SimTime;
         loopUpperBound++) {
      r1->data[r1->size[0] * loopUpperBound] = loopUpperBound;
    }
  }

  /* Time Steps (saved) */
  emxInit_real_T(&r2, 2, true);
  if (rtIsNaN(TimeParams->SimTime)) {
    i0 = r2->size[0] * r2->size[1];
    r2->size[1] = 1;
    emxEnsureCapacity_real_T(r2, i0);
  } else if (TimeParams->SimTime < 0.0) {
    i0 = r2->size[0] * r2->size[1];
    r2->size[1] = 0;
    emxEnsureCapacity_real_T(r2, i0);
  } else if (rtIsInf(TimeParams->SimTime) && (0.0 == TimeParams->SimTime)) {
    i0 = r2->size[0] * r2->size[1];
    r2->size[1] = 1;
    emxEnsureCapacity_real_T(r2, i0);
  } else {
    i0 = r2->size[0] * r2->size[1];
    r2->size[0] = 1;
    r2->size[1] = (int32_T)floor(TimeParams->SimTime / 10.0) + 1;
    emxEnsureCapacity_real_T(r2, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(TimeParams->SimTime /
          10.0); loopUpperBound++) {
      r2->data[r2->size[0] * loopUpperBound] = 10.0 * (real_T)loopUpperBound;
    }
  }

  emxInit_real_T(&EE_mat, 2, true);

  /* Time Steps (saved) */
  /* -------------------------------------------------------------------------- */
  /* Weight Matrices */
  i0 = EE_mat->size[0] * EE_mat->size[1];
  EE_mat->size[0] = (int32_T)PopNum;
  EE_mat->size[1] = (int32_T)PopNum;
  emxEnsureCapacity_real_T(EE_mat, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum * (int32_T)PopNum;
       loopUpperBound++) {
    EE_mat->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T(&II_mat, 2, true);
  i0 = II_mat->size[0] * II_mat->size[1];
  II_mat->size[0] = (int32_T)PopNum;
  II_mat->size[1] = (int32_T)PopNum;
  emxEnsureCapacity_real_T(II_mat, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum * (int32_T)PopNum;
       loopUpperBound++) {
    II_mat->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T(&IE_mat, 2, true);
  i0 = IE_mat->size[0] * IE_mat->size[1];
  IE_mat->size[0] = (int32_T)PopNum;
  IE_mat->size[1] = (int32_T)PopNum;
  emxEnsureCapacity_real_T(IE_mat, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum * (int32_T)PopNum;
       loopUpperBound++) {
    IE_mat->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T(&EI_mat, 2, true);
  i0 = EI_mat->size[0] * EI_mat->size[1];
  EI_mat->size[0] = (int32_T)PopNum;
  EI_mat->size[1] = (int32_T)PopNum;
  emxEnsureCapacity_real_T(EI_mat, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum * (int32_T)PopNum;
       loopUpperBound++) {
    EI_mat->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T(&Ecells, 2, true);
  if (rtIsNaN(PopParams->EPopNum)) {
    i0 = Ecells->size[0] * Ecells->size[1];
    Ecells->size[0] = 1;
    Ecells->size[1] = 1;
    emxEnsureCapacity_real_T(Ecells, i0);
    Ecells->data[0] = rtNaN;
  } else if (PopParams->EPopNum < 1.0) {
    i0 = Ecells->size[0] * Ecells->size[1];
    Ecells->size[0] = 1;
    Ecells->size[1] = 0;
    emxEnsureCapacity_real_T(Ecells, i0);
  } else if (rtIsInf(PopParams->EPopNum) && (1.0 == PopParams->EPopNum)) {
    i0 = Ecells->size[0] * Ecells->size[1];
    Ecells->size[0] = 1;
    Ecells->size[1] = 1;
    emxEnsureCapacity_real_T(Ecells, i0);
    Ecells->data[0] = rtNaN;
  } else {
    i0 = Ecells->size[0] * Ecells->size[1];
    Ecells->size[0] = 1;
    Ecells->size[1] = (int32_T)floor(PopParams->EPopNum - 1.0) + 1;
    emxEnsureCapacity_real_T(Ecells, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(PopParams->EPopNum -
          1.0); loopUpperBound++) {
      Ecells->data[Ecells->size[0] * loopUpperBound] = 1.0 + (real_T)
        loopUpperBound;
    }
  }

  emxInit_real_T(&y, 2, true);
  if (PopNum < 1.0) {
    i0 = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = 0;
    emxEnsureCapacity_real_T(y, i0);
  } else if (rtIsInf(PopNum) && (1.0 == PopNum)) {
    i0 = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = 1;
    emxEnsureCapacity_real_T(y, i0);
    y->data[0] = rtNaN;
  } else {
    i0 = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = (int32_T)floor(PopNum - 1.0) + 1;
    emxEnsureCapacity_real_T(y, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(PopNum - 1.0);
         loopUpperBound++) {
      y->data[y->size[0] * loopUpperBound] = 1.0 + (real_T)loopUpperBound;
    }
  }

  emxInit_boolean_T(&EcellIDX, 2, true);
  i0 = EcellIDX->size[0] * EcellIDX->size[1];
  EcellIDX->size[0] = 1;
  EcellIDX->size[1] = y->size[1];
  emxEnsureCapacity_boolean_T(EcellIDX, i0);
  for (loopUpperBound = 0; loopUpperBound < y->size[0] * y->size[1];
       loopUpperBound++) {
    EcellIDX->data[loopUpperBound] = (y->data[loopUpperBound] <=
      PopParams->EPopNum);
  }

  emxFree_real_T(&y);

  /* EcellIDX = ismember(1:PopNum,Ecells); */
  emxInit_real_T(&Icells, 2, true);
  if (rtIsNaN(PopParams->EPopNum + 1.0)) {
    i0 = Icells->size[0] * Icells->size[1];
    Icells->size[0] = 1;
    Icells->size[1] = 1;
    emxEnsureCapacity_real_T(Icells, i0);
    Icells->data[0] = rtNaN;
  } else if (PopNum < PopParams->EPopNum + 1.0) {
    i0 = Icells->size[0] * Icells->size[1];
    Icells->size[0] = 1;
    Icells->size[1] = 0;
    emxEnsureCapacity_real_T(Icells, i0);
  } else if ((rtIsInf(PopParams->EPopNum + 1.0) || rtIsInf(PopNum)) &&
             (PopParams->EPopNum + 1.0 == PopNum)) {
    i0 = Icells->size[0] * Icells->size[1];
    Icells->size[0] = 1;
    Icells->size[1] = 1;
    emxEnsureCapacity_real_T(Icells, i0);
    Icells->data[0] = rtNaN;
  } else if (floor(PopParams->EPopNum + 1.0) == PopParams->EPopNum + 1.0) {
    ndbl = PopParams->EPopNum + 1.0;
    i0 = Icells->size[0] * Icells->size[1];
    Icells->size[0] = 1;
    Icells->size[1] = (int32_T)floor(PopNum - (PopParams->EPopNum + 1.0)) + 1;
    emxEnsureCapacity_real_T(Icells, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(PopNum - ndbl);
         loopUpperBound++) {
      Icells->data[Icells->size[0] * loopUpperBound] = ndbl + (real_T)
        loopUpperBound;
    }
  } else {
    ndbl = floor((PopNum - (PopParams->EPopNum + 1.0)) + 0.5);
    apnd = (PopParams->EPopNum + 1.0) + ndbl;
    cdiff = apnd - PopNum;
    absa = fabs(PopParams->EPopNum + 1.0);
    if (!((absa > PopNum) || rtIsNaN(PopNum))) {
      absa = PopNum;
    }

    if (fabs(cdiff) < 4.4408920985006262E-16 * absa) {
      ndbl++;
      apnd = PopNum;
    } else if (cdiff > 0.0) {
      apnd = (PopParams->EPopNum + 1.0) + (ndbl - 1.0);
    } else {
      ndbl++;
    }

    if (ndbl >= 0.0) {
      n = (int32_T)ndbl;
    } else {
      n = 0;
    }

    i0 = Icells->size[0] * Icells->size[1];
    Icells->size[0] = 1;
    Icells->size[1] = n;
    emxEnsureCapacity_real_T(Icells, i0);
    if (n > 0) {
      Icells->data[0] = PopParams->EPopNum + 1.0;
      if (n > 1) {
        Icells->data[n - 1] = apnd;
        nm1d2 = (n - 1) / 2;
        for (k = 0; k <= nm1d2 - 2; k++) {
          Icells->data[1 + k] = (EPopNum + 1.0) + (1.0 + (real_T)k);
          Icells->data[(n - k) - 2] = apnd - (1.0 + (real_T)k);
        }

        if (nm1d2 << 1 == n - 1) {
          Icells->data[nm1d2] = ((PopParams->EPopNum + 1.0) + apnd) / 2.0;
        } else {
          Icells->data[nm1d2] = (PopParams->EPopNum + 1.0) + (real_T)nm1d2;
          Icells->data[nm1d2 + 1] = apnd - (real_T)nm1d2;
        }
      }
    }
  }

  emxInit_real_T(&b_y, 2, true);
  if (PopNum < 1.0) {
    i0 = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = 0;
    emxEnsureCapacity_real_T(b_y, i0);
  } else if (rtIsInf(PopNum) && (1.0 == PopNum)) {
    i0 = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = 1;
    emxEnsureCapacity_real_T(b_y, i0);
    b_y->data[0] = rtNaN;
  } else {
    i0 = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = (int32_T)floor(PopNum - 1.0) + 1;
    emxEnsureCapacity_real_T(b_y, i0);
    for (loopUpperBound = 0; loopUpperBound <= (int32_T)(PopNum - 1.0);
         loopUpperBound++) {
      b_y->data[b_y->size[0] * loopUpperBound] = 1.0 + (real_T)loopUpperBound;
    }
  }

  emxInit_boolean_T(&IcellIDX, 2, true);
  i0 = IcellIDX->size[0] * IcellIDX->size[1];
  IcellIDX->size[0] = 1;
  IcellIDX->size[1] = b_y->size[1];
  emxEnsureCapacity_boolean_T(IcellIDX, i0);
  for (loopUpperBound = 0; loopUpperBound < b_y->size[0] * b_y->size[1];
       loopUpperBound++) {
    IcellIDX->data[loopUpperBound] = (b_y->data[loopUpperBound] >
      PopParams->EPopNum);
  }

  emxFree_real_T(&b_y);
  emxInit_int32_T(&r3, 1, true);

  /* IcellIDX = ismember(1:PopNum,Icells); */
  /* Here we assign four 2x2 matrices of matrix (tensor?). There are positive values on the locations where there are connections. */
  /* For example, there are values for the EE connections on the 1x1 matrix, II */
  /* on the 2x2 matrix, and etc (this is based on the indexing of the neuron population).  */
  /* NOTE: presynaptic neurons are columns (dim2) and postsynaptic neurons are rows (dim1). */
  /* E->E Synapses */
  ndbl = PopParams->Kee / (PopParams->EPopNum - 1.0);

  /* -1 to account for self-connections (which are then removed) */
  i0 = r3->size[0];
  r3->size[0] = Ecells->size[1];
  emxEnsureCapacity_int32_T(r3, i0);
  for (loopUpperBound = 0; loopUpperBound < Ecells->size[1]; loopUpperBound++) {
    r3->data[loopUpperBound] = (int32_T)Ecells->data[Ecells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_int32_T(&r4, 1, true);
  i0 = r4->size[0];
  r4->size[0] = Ecells->size[1];
  emxEnsureCapacity_int32_T(r4, i0);
  for (loopUpperBound = 0; loopUpperBound < Ecells->size[1]; loopUpperBound++) {
    r4->data[loopUpperBound] = (int32_T)Ecells->data[Ecells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_real_T(&r, 2, true);
  i0 = r->size[0] * r->size[1];
  r->size[0] = (int32_T)PopParams->EPopNum;
  r->size[1] = (int32_T)PopParams->EPopNum;
  emxEnsureCapacity_real_T(r, i0);
  if (!((r->size[0] == 0) || (r->size[1] == 0))) {
    emlrtRandu(&r->data[0], r->size[0] * r->size[1]);
  }

  for (i0 = 0; i0 < r->size[1]; i0++) {
    for (loopUpperBound = 0; loopUpperBound < r->size[0]; loopUpperBound++) {
      EE_mat->data[r3->data[loopUpperBound] + EE_mat->size[0] * r4->data[i0]] =
        (r->data[loopUpperBound + r->size[0] * i0] <= ndbl);
    }
  }

  emxFree_real_T(&r);
  emxFree_int32_T(&r4);
  emxFree_int32_T(&r3);
  i0 = EE_mat->size[0] * EE_mat->size[1];
  nm1d2 = EE_mat->size[0] * EE_mat->size[1];
  emxEnsureCapacity_real_T(EE_mat, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    EE_mat->data[loopUpperBound] *= PopParams->Wee;
  }

  for (loopUpperBound = 0; loopUpperBound < 2; loopUpperBound++) {
    uv0[loopUpperBound] = (uint32_T)EE_mat->size[loopUpperBound];
  }

  emxInit_boolean_T1(&d, 1, true);
  if (((int32_T)uv0[0] == 1) && ((int32_T)uv0[1] == 1)) {
    i0 = d->size[0];
    d->size[0] = 1;
    emxEnsureCapacity_boolean_T1(d, i0);
  } else {
    if (0 < (int32_T)uv0[1]) {
      if ((int32_T)uv0[0] < (int32_T)uv0[1]) {
        nm1d2 = (int32_T)uv0[0];
      } else {
        nm1d2 = (int32_T)uv0[1];
      }
    } else {
      nm1d2 = 0;
    }

    i0 = d->size[0];
    d->size[0] = nm1d2;
    emxEnsureCapacity_boolean_T1(d, i0);
    for (ii = 0; ii < nm1d2; ii++) {
      d->data[ii] = true;
    }
  }

  emxInit_boolean_T(&r5, 2, true);
  iv0[0] = d->size[0];
  iv0[1] = d->size[0];
  i0 = r5->size[0] * r5->size[1];
  r5->size[0] = iv0[0];
  r5->size[1] = iv0[1];
  emxEnsureCapacity_boolean_T(r5, i0);
  for (loopUpperBound = 0; loopUpperBound < iv0[0] * iv0[1]; loopUpperBound++) {
    r5->data[loopUpperBound] = false;
  }

  b_loopUpperBound = d->size[0];
  emxFree_boolean_T(&d);
  for (ii = 0; ii < b_loopUpperBound; ii++) {
    r5->data[ii + r5->size[0] * ii] = true;
  }

  n = r5->size[0] * r5->size[1] - 1;
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (r5->data[mA]) {
      nm1d2++;
    }
  }

  emxInit_int32_T(&r6, 1, true);
  i0 = r6->size[0];
  r6->size[0] = nm1d2;
  emxEnsureCapacity_int32_T(r6, i0);
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (r5->data[mA]) {
      r6->data[nm1d2] = mA + 1;
      nm1d2++;
    }
  }

  emxFree_boolean_T(&r5);
  i0 = r6->size[0];
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    EE_mat->data[r6->data[loopUpperBound] - 1] = 0.0;
  }

  emxFree_int32_T(&r6);
  emxInit_int32_T(&r7, 1, true);

  /* Remove selfconnections */
  /* I->I Synapses */
  ndbl = PopParams->Kii / (PopParams->IPopNum - 1.0);
  i0 = r7->size[0];
  r7->size[0] = Icells->size[1];
  emxEnsureCapacity_int32_T(r7, i0);
  for (loopUpperBound = 0; loopUpperBound < Icells->size[1]; loopUpperBound++) {
    r7->data[loopUpperBound] = (int32_T)Icells->data[Icells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_int32_T(&r8, 1, true);
  i0 = r8->size[0];
  r8->size[0] = Icells->size[1];
  emxEnsureCapacity_int32_T(r8, i0);
  for (loopUpperBound = 0; loopUpperBound < Icells->size[1]; loopUpperBound++) {
    r8->data[loopUpperBound] = (int32_T)Icells->data[Icells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_real_T(&b_r, 2, true);
  i0 = b_r->size[0] * b_r->size[1];
  b_r->size[0] = (int32_T)PopParams->IPopNum;
  b_r->size[1] = (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T(b_r, i0);
  if (!((b_r->size[0] == 0) || (b_r->size[1] == 0))) {
    emlrtRandu(&b_r->data[0], b_r->size[0] * b_r->size[1]);
  }

  for (i0 = 0; i0 < b_r->size[1]; i0++) {
    for (loopUpperBound = 0; loopUpperBound < b_r->size[0]; loopUpperBound++) {
      II_mat->data[r7->data[loopUpperBound] + II_mat->size[0] * r8->data[i0]] =
        (b_r->data[loopUpperBound + b_r->size[0] * i0] <= ndbl);
    }
  }

  emxFree_real_T(&b_r);
  emxFree_int32_T(&r8);
  emxFree_int32_T(&r7);
  i0 = II_mat->size[0] * II_mat->size[1];
  nm1d2 = II_mat->size[0] * II_mat->size[1];
  emxEnsureCapacity_real_T(II_mat, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    II_mat->data[loopUpperBound] *= PopParams->Wii;
  }

  for (loopUpperBound = 0; loopUpperBound < 2; loopUpperBound++) {
    uv0[loopUpperBound] = (uint32_T)II_mat->size[loopUpperBound];
  }

  emxInit_boolean_T1(&b_d, 1, true);
  if (((int32_T)uv0[0] == 1) && ((int32_T)uv0[1] == 1)) {
    i0 = b_d->size[0];
    b_d->size[0] = 1;
    emxEnsureCapacity_boolean_T1(b_d, i0);
  } else {
    if (0 < (int32_T)uv0[1]) {
      if ((int32_T)uv0[0] < (int32_T)uv0[1]) {
        nm1d2 = (int32_T)uv0[0];
      } else {
        nm1d2 = (int32_T)uv0[1];
      }
    } else {
      nm1d2 = 0;
    }

    i0 = b_d->size[0];
    b_d->size[0] = nm1d2;
    emxEnsureCapacity_boolean_T1(b_d, i0);
    for (ii = 0; ii < nm1d2; ii++) {
      b_d->data[ii] = true;
    }
  }

  emxInit_boolean_T(&r9, 2, true);
  iv0[0] = b_d->size[0];
  iv0[1] = b_d->size[0];
  i0 = r9->size[0] * r9->size[1];
  r9->size[0] = iv0[0];
  r9->size[1] = iv0[1];
  emxEnsureCapacity_boolean_T(r9, i0);
  for (loopUpperBound = 0; loopUpperBound < iv0[0] * iv0[1]; loopUpperBound++) {
    r9->data[loopUpperBound] = false;
  }

  b_loopUpperBound = b_d->size[0];
  emxFree_boolean_T(&b_d);
  for (ii = 0; ii < b_loopUpperBound; ii++) {
    r9->data[ii + r9->size[0] * ii] = true;
  }

  n = r9->size[0] * r9->size[1] - 1;
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (r9->data[mA]) {
      nm1d2++;
    }
  }

  emxInit_int32_T(&r10, 1, true);
  i0 = r10->size[0];
  r10->size[0] = nm1d2;
  emxEnsureCapacity_int32_T(r10, i0);
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (r9->data[mA]) {
      r10->data[nm1d2] = mA + 1;
      nm1d2++;
    }
  }

  emxFree_boolean_T(&r9);
  i0 = r10->size[0];
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    II_mat->data[r10->data[loopUpperBound] - 1] = 0.0;
  }

  emxFree_int32_T(&r10);
  emxInit_int32_T(&r11, 1, true);

  /* Remove selfconnections */
  /* E->I Synapses */
  ndbl = PopParams->Kie / PopParams->EPopNum;
  i0 = r11->size[0];
  r11->size[0] = Icells->size[1];
  emxEnsureCapacity_int32_T(r11, i0);
  for (loopUpperBound = 0; loopUpperBound < Icells->size[1]; loopUpperBound++) {
    r11->data[loopUpperBound] = (int32_T)Icells->data[Icells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_int32_T(&r12, 1, true);
  i0 = r12->size[0];
  r12->size[0] = Ecells->size[1];
  emxEnsureCapacity_int32_T(r12, i0);
  for (loopUpperBound = 0; loopUpperBound < Ecells->size[1]; loopUpperBound++) {
    r12->data[loopUpperBound] = (int32_T)Ecells->data[Ecells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_real_T(&c_r, 2, true);
  i0 = c_r->size[0] * c_r->size[1];
  c_r->size[0] = (int32_T)PopParams->IPopNum;
  c_r->size[1] = (int32_T)PopParams->EPopNum;
  emxEnsureCapacity_real_T(c_r, i0);
  if (!((c_r->size[0] == 0) || (c_r->size[1] == 0))) {
    emlrtRandu(&c_r->data[0], c_r->size[0] * c_r->size[1]);
  }

  for (i0 = 0; i0 < c_r->size[1]; i0++) {
    for (loopUpperBound = 0; loopUpperBound < c_r->size[0]; loopUpperBound++) {
      IE_mat->data[r11->data[loopUpperBound] + IE_mat->size[0] * r12->data[i0]] =
        (c_r->data[loopUpperBound + c_r->size[0] * i0] <= ndbl);
    }
  }

  emxFree_real_T(&c_r);
  emxFree_int32_T(&r12);
  emxFree_int32_T(&r11);
  i0 = IE_mat->size[0] * IE_mat->size[1];
  nm1d2 = IE_mat->size[0] * IE_mat->size[1];
  emxEnsureCapacity_real_T(IE_mat, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    IE_mat->data[loopUpperBound] *= PopParams->Wie;
  }

  emxInit_int32_T(&r13, 1, true);

  /* I->E Synapses */
  ndbl = PopParams->Kei / PopParams->IPopNum;
  i0 = r13->size[0];
  r13->size[0] = Ecells->size[1];
  emxEnsureCapacity_int32_T(r13, i0);
  for (loopUpperBound = 0; loopUpperBound < Ecells->size[1]; loopUpperBound++) {
    r13->data[loopUpperBound] = (int32_T)Ecells->data[Ecells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_int32_T(&r14, 1, true);
  i0 = r14->size[0];
  r14->size[0] = Icells->size[1];
  emxEnsureCapacity_int32_T(r14, i0);
  for (loopUpperBound = 0; loopUpperBound < Icells->size[1]; loopUpperBound++) {
    r14->data[loopUpperBound] = (int32_T)Icells->data[Icells->size[0] *
      loopUpperBound] - 1;
  }

  emxInit_real_T(&d_r, 2, true);
  i0 = d_r->size[0] * d_r->size[1];
  d_r->size[0] = (int32_T)PopParams->EPopNum;
  d_r->size[1] = (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T(d_r, i0);
  if (!((d_r->size[0] == 0) || (d_r->size[1] == 0))) {
    emlrtRandu(&d_r->data[0], d_r->size[0] * d_r->size[1]);
  }

  for (i0 = 0; i0 < d_r->size[1]; i0++) {
    for (loopUpperBound = 0; loopUpperBound < d_r->size[0]; loopUpperBound++) {
      EI_mat->data[r13->data[loopUpperBound] + EI_mat->size[0] * r14->data[i0]] =
        (d_r->data[loopUpperBound + d_r->size[0] * i0] <= ndbl);
    }
  }

  emxFree_real_T(&d_r);
  emxFree_int32_T(&r14);
  emxFree_int32_T(&r13);
  i0 = EI_mat->size[0] * EI_mat->size[1];
  nm1d2 = EI_mat->size[0] * EI_mat->size[1];
  emxEnsureCapacity_real_T(EI_mat, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    EI_mat->data[loopUpperBound] *= PopParams->Wei;
  }

  emxInit_boolean_T(&isconnected, 2, true);
  i0 = isconnected->size[0] * isconnected->size[1];
  isconnected->size[0] = EE_mat->size[0];
  isconnected->size[1] = EE_mat->size[1];
  emxEnsureCapacity_boolean_T(isconnected, i0);
  for (loopUpperBound = 0; loopUpperBound < EE_mat->size[0] * EE_mat->size[1];
       loopUpperBound++) {
    isconnected->data[loopUpperBound] = (((EE_mat->data[loopUpperBound] +
      II_mat->data[loopUpperBound]) + EI_mat->data[loopUpperBound]) +
      IE_mat->data[loopUpperBound] > 0.0);
  }

  emxInit_real_T1(&E_L, 1, true);

  /*  */
  /* -------------------------------------------------------------------------- */
  /* Simulation Parameters */
  /* LIF Parameters */
  /*  E_L         = PopParams.E_L;      %Reversal potential (mV) */
  /*  g_L         = PopParams.g_L;     %conductance (units?) */
  /*  C           = PopParams.C;       %capacitance (nF) */
  /*  I_e         = PopParams.I_e;      %current (nA) */
  /*  V_th        = PopParams.V_th;    %spike threshhold (mV) */
  /*  V_reset     = PopParams.V_reset; %reset value (mV) */
  /*   */
  /*  t_ref       = PopParams.t_ref;   %refractory period (ms) */
  /*   */
  /*  sigma       = PopParams.sigma;   %Standard deviation of noise */
  /*  theta       = PopParams.theta;   %Strength to mean (time scale of noise, ms^-1) */
  /*   */
  /*  %-------------------------------------------------------------------------- */
  /*  %Adaptation */
  /*  E_w         = PopParams.E_w;     %Adaptation reversal potential, (mV) */
  /*  b_w         = PopParams.b_w;     %Adaptation decay (1/ms) */
  /*   */
  /*  delta_T     = PopParams.delta_T; %threshhold softness */
  /*   */
  /*  gwnorm      = PopParams.gwnorm;  %Adaptation norm (nS) */
  /*  w_r         = PopParams.w_r;     %Adaptation at rest (E_L) */
  /*   */
  /*  b           = PopParams.b;       %Spike Adaptation (nS) */
  /*   */
  /*  %-------------------------------------------------------------------------- */
  /*  % Synapse Parameters */
  /*  E_e         = PopParams.E_e;     %Excitatory reversal potential (mV) */
  /*  E_i         = PopParams.E_i;     %Inhibitory reversal potential (mV) */
  /*   */
  /*  tau_s       = PopParams.tau_s;     %Synaptic decay (ms) */
  /*   */
  /*  %STDP parameters */
  /*  LearningRate = PopParams.LearningRate; */
  /*  TargetRate   = PopParams.TargetRate; %Target Rate for Excitatory cells (units of Hz) */
  /*  tauSTDP      = PopParams.tauSTDP;    %Time Constant for the STDP curve (Units of ms) */
  /*  alpha = 2.*(PopParams.TargetRate./1000).*tauSTDP; %Alpha parameter from Vogels eqn5 */
  /* Note target rate is converted to spks/ms */
  i0 = E_L->size[0];
  E_L->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(E_L, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    E_L->data[loopUpperBound] = PopParams->E_L[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    E_L->data[loopUpperBound + (int32_T)PopParams->EPopNum] = PopParams->E_L[1];
  }

  emxInit_real_T1(&g_L, 1, true);
  i0 = g_L->size[0];
  g_L->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(g_L, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    g_L->data[loopUpperBound] = PopParams->g_L[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    g_L->data[loopUpperBound + (int32_T)PopParams->EPopNum] = PopParams->g_L[1];
  }

  emxInit_real_T1(&C, 1, true);
  i0 = C->size[0];
  C->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(C, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    C->data[loopUpperBound] = PopParams->C[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    C->data[loopUpperBound + (int32_T)PopParams->EPopNum] = PopParams->C[1];
  }

  emxInit_real_T1(&I_e, 1, true);
  i0 = I_e->size[0];
  I_e->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(I_e, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    I_e->data[loopUpperBound] = PopParams->I_e;
  }

  emxInit_real_T1(&V_th, 1, true);
  i0 = V_th->size[0];
  V_th->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(V_th, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    V_th->data[loopUpperBound] = PopParams->V_th[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    V_th->data[loopUpperBound + (int32_T)PopParams->EPopNum] = PopParams->V_th[1];
  }

  emxInit_real_T1(&V_reset, 1, true);
  i0 = V_reset->size[0];
  V_reset->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(V_reset, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    V_reset->data[loopUpperBound] = PopParams->V_reset[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    V_reset->data[loopUpperBound + (int32_T)PopParams->EPopNum] =
      PopParams->V_reset[1];
  }

  emxInit_real_T1(&t_ref, 1, true);
  i0 = t_ref->size[0];
  t_ref->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(t_ref, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    t_ref->data[loopUpperBound] = PopParams->t_ref;
  }

  emxInit_real_T1(&sigma, 1, true);
  i0 = sigma->size[0];
  sigma->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(sigma, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    sigma->data[loopUpperBound] = PopParams->sigma;
  }

  emxInit_real_T1(&theta, 1, true);
  i0 = theta->size[0];
  theta->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(theta, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    theta->data[loopUpperBound] = PopParams->theta;
  }

  emxInit_real_T1(&E_w, 1, true);
  i0 = E_w->size[0];
  E_w->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(E_w, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    E_w->data[loopUpperBound] = PopParams->E_w;
  }

  emxInit_real_T1(&b_w, 1, true);
  i0 = b_w->size[0];
  b_w->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(b_w, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    b_w->data[loopUpperBound] = PopParams->b_w;
  }

  emxInit_real_T1(&delta_T, 1, true);
  i0 = delta_T->size[0];
  delta_T->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(delta_T, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    delta_T->data[loopUpperBound] = PopParams->delta_T;
  }

  emxInit_real_T1(&gwnorm, 1, true);
  i0 = gwnorm->size[0];
  gwnorm->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(gwnorm, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    gwnorm->data[loopUpperBound] = PopParams->gwnorm;
  }

  emxInit_real_T1(&w_r, 1, true);
  i0 = w_r->size[0];
  w_r->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(w_r, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    w_r->data[loopUpperBound] = PopParams->w_r;
  }

  emxInit_real_T1(&b, 1, true);
  i0 = b->size[0];
  b->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(b, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    b->data[loopUpperBound] = PopParams->b;
  }

  emxInit_real_T1(&E_e, 1, true);
  i0 = E_e->size[0];
  E_e->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(E_e, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    E_e->data[loopUpperBound] = PopParams->E_e;
  }

  emxInit_real_T1(&E_i, 1, true);
  i0 = E_i->size[0];
  E_i->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(E_i, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    E_i->data[loopUpperBound] = PopParams->E_i;
  }

  emxInit_real_T1(&tau_s, 1, true);
  i0 = tau_s->size[0];
  tau_s->size[0] = (int32_T)PopParams->EPopNum + (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T1(tau_s, i0);
  nm1d2 = (int32_T)PopParams->EPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    tau_s->data[loopUpperBound] = PopParams->tau_s[0];
  }

  nm1d2 = (int32_T)PopParams->IPopNum - 1;
  for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
    tau_s->data[loopUpperBound + (int32_T)PopParams->EPopNum] = PopParams->
      tau_s[1];
  }

  emxInit_real_T1(&ELearningRate, 1, true);
  i0 = ELearningRate->size[0];
  ELearningRate->size[0] = (int32_T)PopParams->EPopNum;
  emxEnsureCapacity_real_T1(ELearningRate, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopParams->EPopNum;
       loopUpperBound++) {
    ELearningRate->data[loopUpperBound] = PopParams->LearningRate;
  }

  emxInit_real_T(&ILearningRate, 2, true);
  i0 = ILearningRate->size[0] * ILearningRate->size[1];
  ILearningRate->size[0] = 1;
  ILearningRate->size[1] = (int32_T)PopParams->IPopNum;
  emxEnsureCapacity_real_T(ILearningRate, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopParams->IPopNum;
       loopUpperBound++) {
    ILearningRate->data[loopUpperBound] = PopParams->LearningRate;
  }

  emxInit_real_T1(&tauSTDP, 1, true);

  /* Target Rate for Excitatory cells (units of Hz) */
  i0 = tauSTDP->size[0];
  tauSTDP->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(tauSTDP, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    tauSTDP->data[loopUpperBound] = PopParams->tauSTDP;
  }

  emxInit_real_T1(&alpha, 1, true);

  /* Time Constant for the STDP curve (Units of ms) */
  ndbl = 2.0 * (PopParams->TargetRate / 1000.0) * PopParams->tauSTDP;
  i0 = alpha->size[0];
  alpha->size[0] = (int32_T)PopParams->EPopNum;
  emxEnsureCapacity_real_T1(alpha, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopParams->EPopNum;
       loopUpperBound++) {
    alpha->data[loopUpperBound] = ndbl;
  }

  emxInit_real_T1(&V, 1, true);

  /* Alpha parameter from Vogels eqn5 */
  /*  Input: convert into function of t */
  /*  if isa(I_e, 'function_handle') */
  /*  elseif isequal(size(I_e),[1 1]) */
  /*      I_e = @(t) I_e; */
  /*  elseif length(I_e) == 2 */
  /*      I_e = @(t) transpose([I_e(1).*ones(1,EPopNum),     I_e(2).*ones(1,IPopNum)]); */
  /*  end */
  /*  Variables */
  /* Simulation Variables */
  i0 = V->size[0];
  V->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(V, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    V->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&g_e, 1, true);

  /* Membrane Potential */
  i0 = g_e->size[0];
  g_e->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(g_e, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    g_e->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&g_i, 1, true);

  /* conductance of synapse  */
  i0 = g_i->size[0];
  g_i->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(g_i, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    g_i->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&g_w, 1, true);

  /* conductance of synapse  */
  i0 = g_w->size[0];
  g_w->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(g_w, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    g_w->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&a_w, 1, true);

  /* conductance of adaptation  */
  i0 = a_w->size[0];
  a_w->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(a_w, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    a_w->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&s, 1, true);

  /* adaptation rise (1/ms) */
  i0 = s->size[0];
  s->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(s, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    s->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&w, 1, true);

  /* synapse  */
  i0 = w->size[0];
  w->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(w, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    w->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&X_t, 1, true);

  /* adaptation */
  i0 = X_t->size[0];
  X_t->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(X_t, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    X_t->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&t_r, 1, true);

  /* OU noise */
  i0 = t_r->size[0];
  t_r->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(t_r, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    t_r->data[loopUpperBound] = 0.0;
  }

  emxInit_real_T1(&x, 1, true);
  i0 = x->size[0];
  x->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(x, i0);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)PopNum; loopUpperBound++) {
    x->data[loopUpperBound] = 0.0;
  }

  /* Synaptic trace */
  /* Saved Variables */
  i0 = SimValues->t->size[0] * SimValues->t->size[1];
  SimValues->t->size[0] = 1;
  SimValues->t->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->t, i0);
  for (i0 = 0; i0 < r1->size[1]; i0++) {
    SimValues->t->data[i0] = rtNaN;
  }

  i0 = SimValues->V->size[0] * SimValues->V->size[1];
  SimValues->V->size[0] = (int32_T)PopNum;
  SimValues->V->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->V, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->V->data[i0] = rtNaN;
  }

  i0 = SimValues->g_w->size[0] * SimValues->g_w->size[1];
  SimValues->g_w->size[0] = (int32_T)PopNum;
  SimValues->g_w->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->g_w, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->g_w->data[i0] = rtNaN;
  }

  i0 = SimValues->g_e->size[0] * SimValues->g_e->size[1];
  SimValues->g_e->size[0] = (int32_T)PopNum;
  SimValues->g_e->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->g_e, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->g_e->data[i0] = rtNaN;
  }

  i0 = SimValues->g_i->size[0] * SimValues->g_i->size[1];
  SimValues->g_i->size[0] = (int32_T)PopNum;
  SimValues->g_i->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->g_i, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->g_i->data[i0] = rtNaN;
  }

  i0 = SimValues->s->size[0] * SimValues->s->size[1];
  SimValues->s->size[0] = (int32_T)PopNum;
  SimValues->s->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->s, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->s->data[i0] = rtNaN;
  }

  i0 = SimValues->w->size[0] * SimValues->w->size[1];
  SimValues->w->size[0] = (int32_T)PopNum;
  SimValues->w->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->w, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->w->data[i0] = rtNaN;
  }

  i0 = SimValues->a_w->size[0] * SimValues->a_w->size[1];
  SimValues->a_w->size[0] = (int32_T)PopNum;
  SimValues->a_w->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->a_w, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->a_w->data[i0] = rtNaN;
  }

  i0 = SimValues->Input->size[0] * SimValues->Input->size[1];
  SimValues->Input->size[0] = (int32_T)PopNum;
  SimValues->Input->size[1] = r1->size[1];
  emxEnsureCapacity_real_T(SimValues->Input, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * r1->size[1]; i0++) {
    SimValues->Input->data[i0] = rtNaN;
  }

  emxFree_real_T(&r1);
  i0 = SimValues->t_weight->size[0] * SimValues->t_weight->size[1];
  SimValues->t_weight->size[0] = 1;
  SimValues->t_weight->size[1] = r2->size[1];
  emxEnsureCapacity_real_T(SimValues->t_weight, i0);
  for (i0 = 0; i0 < r2->size[1]; i0++) {
    SimValues->t_weight->data[i0] = rtNaN;
  }

  i0 = SimValues->WeightMat->size[0] * SimValues->WeightMat->size[1] *
    SimValues->WeightMat->size[2];
  SimValues->WeightMat->size[0] = (int32_T)PopNum;
  SimValues->WeightMat->size[1] = (int32_T)PopNum;
  SimValues->WeightMat->size[2] = r2->size[1];
  emxEnsureCapacity_real_T2(SimValues->WeightMat, i0);
  for (i0 = 0; i0 < (int32_T)PopNum * (int32_T)PopNum * r2->size[1]; i0++) {
    SimValues->WeightMat->data[i0] = rtNaN;
  }

  emxFree_real_T(&r2);
  emxInit_boolean_T(&b_x, 2, true);
  i0 = b_x->size[0] * b_x->size[1];
  b_x->size[0] = 1;
  b_x->size[1] = recordVALs->size[1];
  emxEnsureCapacity_boolean_T(b_x, i0);
  for (loopUpperBound = 0; loopUpperBound < recordVALs->size[0] *
       recordVALs->size[1]; loopUpperBound++) {
    b_x->data[loopUpperBound] = (recordVALs->data[loopUpperBound] != 0);
  }

  emxInit_int32_T1(&b_ii, 2, true);
  nx = b_x->size[1];
  idx = 0;
  i0 = b_ii->size[0] * b_ii->size[1];
  b_ii->size[0] = 1;
  b_ii->size[1] = b_x->size[1];
  emxEnsureCapacity_int32_T1(b_ii, i0);
  nm1d2 = 1;
  exitg1 = false;
  while ((!exitg1) && (nm1d2 <= nx)) {
    if (b_x->data[nm1d2 - 1]) {
      idx++;
      b_ii->data[idx - 1] = nm1d2;
      if (idx >= nx) {
        exitg1 = true;
      } else {
        nm1d2++;
      }
    } else {
      nm1d2++;
    }
  }

  if (b_x->size[1] == 1) {
    if (idx == 0) {
      i0 = b_ii->size[0] * b_ii->size[1];
      b_ii->size[1] = 0;
      emxEnsureCapacity_int32_T1(b_ii, i0);
    }
  } else {
    i0 = b_ii->size[0] * b_ii->size[1];
    if (1 > idx) {
      b_ii->size[1] = 0;
    } else {
      b_ii->size[1] = idx;
    }

    emxEnsureCapacity_int32_T1(b_ii, i0);
  }

  emxFree_boolean_T(&b_x);
  emxInit_real32_T(&spikes, 2, true);
  ndbl = rt_roundd_snf((real_T)b_ii->size[1] * TimeParams->dt);
  i0 = spikes->size[0] * spikes->size[1];
  spikes->size[0] = (int32_T)(PopNum * ndbl * 20.0);
  spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(spikes, i0);
  emxFree_int32_T(&b_ii);
  for (loopUpperBound = 0; loopUpperBound < (int32_T)(PopNum * ndbl * 20.0) * 2;
       loopUpperBound++) {
    spikes->data[loopUpperBound] = ((real32_T)rtNaN);
  }

  /* assume mean rate 20Hz */
  /*  if no spike adaptation, set to steady state????? or set to alpha(v_th) */
  n = b->size[0] - 1;
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (b->data[mA] == 0.0) {
      nm1d2++;
    }
  }

  emxInit_int32_T(&r15, 1, true);
  i0 = r15->size[0];
  r15->size[0] = nm1d2;
  emxEnsureCapacity_int32_T(r15, i0);
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (b->data[mA] == 0.0) {
      r15->data[nm1d2] = mA + 1;
      nm1d2++;
    }
  }

  n = b->size[0] - 1;
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (b->data[mA] == 0.0) {
      nm1d2++;
    }
  }

  emxInit_int32_T(&r16, 1, true);
  i0 = r16->size[0];
  r16->size[0] = nm1d2;
  emxEnsureCapacity_int32_T(r16, i0);
  nm1d2 = 0;
  for (mA = 0; mA <= n; mA++) {
    if (b->data[mA] == 0.0) {
      r16->data[nm1d2] = mA + 1;
      nm1d2++;
    }
  }

  emxInit_real_T1(&c_x, 1, true);
  i0 = c_x->size[0];
  c_x->size[0] = r16->size[0];
  emxEnsureCapacity_real_T1(c_x, i0);
  for (loopUpperBound = 0; loopUpperBound < r16->size[0]; loopUpperBound++) {
    c_x->data[loopUpperBound] = (V_reset->data[r16->data[loopUpperBound] - 1] -
      E_L->data[r16->data[loopUpperBound] - 1]) * delta_T->data[r16->
      data[loopUpperBound] - 1];
  }

  emxFree_int32_T(&r16);
  nx = c_x->size[0];
  for (ii = 0; ii < nx; ii++) {
    c_x->data[ii] = exp(c_x->data[ii]);
  }

  emxInit_real_T1(&r17, 1, true);
  i0 = r17->size[0];
  r17->size[0] = r15->size[0];
  emxEnsureCapacity_real_T1(r17, i0);
  for (loopUpperBound = 0; loopUpperBound < r15->size[0]; loopUpperBound++) {
    r17->data[loopUpperBound] = w_r->data[r15->data[loopUpperBound] - 1] *
      b_w->data[r15->data[loopUpperBound] - 1] / (1.0 - w_r->data[r15->
      data[loopUpperBound] - 1]) * c_x->data[loopUpperBound];
  }

  emxFree_real_T(&c_x);
  emxFree_int32_T(&r15);
  n = b->size[0];
  nm1d2 = 0;
  for (mA = 0; mA < n; mA++) {
    if (b->data[mA] == 0.0) {
      b->data[mA] = r17->data[nm1d2];
      nm1d2++;
    }
  }

  emxFree_real_T(&r17);
  emxInit_real_T1(&dX, 1, true);

  /*  If no noise */
  i0 = dX->size[0];
  dX->size[0] = 1;
  emxEnsureCapacity_real_T1(dX, i0);
  dX->data[0] = 0.0;

  /* V - Voltage Equation */
  /* s - Synaptic Output  */
  /* w - Adaptation Variable */
  /* x - Synaptic Trace for STDP */
  /*  Initial Conditions - random voltages */
  /* Improvement?: set # initial spiking neurons instead of hard coding  */
  /* range: E_L-Vth */
  /* chance of initial spiking  */
  cdiff = E_L->data[0];
  b_loopUpperBound = E_L->size[0];
  for (mA = 0; mA <= b_loopUpperBound - 2; mA++) {
    p = ((!rtIsNaN(E_L->data[mA + 1])) && (rtIsNaN(cdiff) || (cdiff > E_L->
           data[mA + 1])));
    if (p) {
      cdiff = E_L->data[mA + 1];
    }
  }

  ndbl = V_th->data[0];
  b_loopUpperBound = V_th->size[0];
  for (mA = 0; mA <= b_loopUpperBound - 2; mA++) {
    p = ((!rtIsNaN(V_th->data[mA + 1])) && (rtIsNaN(ndbl) || (ndbl < V_th->
           data[mA + 1])));
    if (p) {
      ndbl = V_th->data[mA + 1];
    }
  }

  emxInit_real_T1(&e_r, 1, true);

  /* make this neuron vector */
  ndbl -= cdiff;
  i0 = e_r->size[0];
  e_r->size[0] = (int32_T)PopNum;
  emxEnsureCapacity_real_T1(e_r, i0);
  if (!(e_r->size[0] == 0)) {
    emlrtRandu(&e_r->data[0], e_r->size[0]);
  }

  for (loopUpperBound = 0; loopUpperBound < e_r->size[0]; loopUpperBound++) {
    V->data[loopUpperBound] = cdiff + ndbl * e_r->data[loopUpperBound];
  }

  emxFree_real_T(&e_r);

  /*  Time Loop */
  savecounter = 1U;
  weightcounter = 1U;
  cdiff = 0.0;
  b_loopUpperBound = r0->size[1] - 1;
  emxFree_real_T(&r0);
  emxInit_real_T1(&PreIspikes, 1, true);
  emxInit_real_T1(&PostEspikes, 1, true);
  emxInit_real_T1(&r18, 1, true);
  emxInit_real_T1(&r19, 1, true);
  emxInit_real_T(&r20, 2, true);
  emxInit_int32_T(&r21, 1, true);
  emxInit_int32_T1(&r22, 2, true);
  emxInit_int32_T1(&r23, 2, true);
  emxInit_int32_T1(&r24, 2, true);
  emxInit_int32_T1(&r25, 2, true);
  emxInit_int32_T(&r26, 1, true);
  emxInit_int32_T(&r27, 1, true);
  emxInit_boolean_T1(&d_x, 1, true);
  emxInit_real_T1(&e_x, 1, true);
  emxInit_real_T1(&f_x, 1, true);
  emxInit_real_T1(&g_x, 1, true);
  emxInit_boolean_T1(&h_x, 1, true);
  emxInit_boolean_T1(&i_x, 1, true);
  emxInit_int32_T(&c_ii, 1, true);
  emxInitMatrix_cell_wrap_1(reshapes, true);
  emxInit_boolean_T1(&j_x, 1, true);
  emxInit_real_T1(&b_C, 1, true);
  emxInit_real_T1(&c_C, 1, true);
  emxInit_real_T1(&d_C, 1, true);
  emxInit_real_T1(&e_C, 1, true);
  emxInit_real_T1(&b_s, 1, true);
  emxInit_real_T1(&k_x, 1, true);
  emxInit_real_T(&b_EI_mat, 2, true);
  emxInit_real_T(&c_EI_mat, 2, true);
  emxInit_real_T1(&b_t_r, 1, true);
  if (0 <= b_loopUpperBound) {
    l_x = (int32_T)rt_roundd_snf((real_T)varargin_2 / 10.0);
    if ((!rtIsInf(-0.0 - TimeParams->dt)) && (!rtIsNaN(-0.0 - TimeParams->dt)))
    {
      if (-0.0 - TimeParams->dt == 0.0) {
        f_r = 0.0;
      } else {
        f_r = fmod(-0.0 - TimeParams->dt, 1.0);
        if (f_r == 0.0) {
          f_r = 0.0;
        } else {
          if (-0.0 - TimeParams->dt < 0.0) {
            f_r++;
          }
        }
      }

      if (-0.0 - TimeParams->dt == 0.0) {
        g_r = 0.0;
      } else {
        g_r = fmod(-0.0 - TimeParams->dt, 10.0);
        if (g_r == 0.0) {
          g_r = 0.0;
        } else {
          if (-0.0 - TimeParams->dt < 0.0) {
            g_r += 10.0;
          }
        }
      }
    } else {
      f_r = rtNaN;
      g_r = rtNaN;
    }
  }

  for (tt = 0; tt <= b_loopUpperBound; tt++) {
    /*     %% Time Counter */
    nm1d2 = 1 + tt;
    if (l_x != 0) {
      nm1d2 = (int32_T)fmod(1.0 + (real_T)tt, l_x);
    }

    if (nm1d2 == 0) {
      /* clearly, this needs improvement */
      c_y = NULL;
      m0 = emlrtCreateDoubleScalar(rt_roundd_snf(100.0 * (1.0 + (real_T)tt) /
        (real_T)varargin_2));
      emlrtAssign(&c_y, m0);
      emlrtDisplayR2012b(c_y, "ans", &emlrtRTEI, emlrtRootTLSGlobal);
    }

    /*     %% Dynamics: update noise, V,s,w based on values in previous timestep */
    /* Noise input (independent for each neuron... could also be correlated) */
    /* To do: precompte drive, so don't need random number generation each dt */
    i0 = d_x->size[0];
    d_x->size[0] = sigma->size[0];
    emxEnsureCapacity_boolean_T1(d_x, i0);
    for (loopUpperBound = 0; loopUpperBound < sigma->size[0]; loopUpperBound++)
    {
      d_x->data[loopUpperBound] = (sigma->data[loopUpperBound] != 0.0);
    }

    p = !(d_x->size[0] == 0);
    if (p) {
      k = 1;
      exitg1 = false;
      while ((!exitg1) && (k <= d_x->size[0])) {
        if (!d_x->data[k - 1]) {
          p = false;
          exitg1 = true;
        } else {
          k++;
        }
      }
    }

    if (p) {
      i0 = e_x->size[0];
      e_x->size[0] = theta->size[0];
      emxEnsureCapacity_real_T1(e_x, i0);
      for (loopUpperBound = 0; loopUpperBound < theta->size[0]; loopUpperBound++)
      {
        e_x->data[loopUpperBound] = 2.0 * theta->data[loopUpperBound];
      }

      nx = e_x->size[0];
      for (ii = 0; ii < nx; ii++) {
        e_x->data[ii] = sqrt(e_x->data[ii]);
      }

      i0 = r18->size[0];
      r18->size[0] = (int32_T)PopNum;
      emxEnsureCapacity_real_T1(r18, i0);
      if (!(r18->size[0] == 0)) {
        emlrtRandn(&r18->data[0], r18->size[0]);
      }

      ndbl = sqrt(dt);
      i0 = dX->size[0];
      dX->size[0] = theta->size[0];
      emxEnsureCapacity_real_T1(dX, i0);
      for (loopUpperBound = 0; loopUpperBound < theta->size[0]; loopUpperBound++)
      {
        dX->data[loopUpperBound] = -theta->data[loopUpperBound] * X_t->
          data[loopUpperBound] * dt + e_x->data[loopUpperBound] * sigma->
          data[loopUpperBound] * r18->data[loopUpperBound] * ndbl;
      }
    }

    /* V - Voltage Equation */
    /*                       %Leak */
    /*                       %Adaptation */
    /*        %Synapses */
    i0 = f_x->size[0];
    f_x->size[0] = g_L->size[0];
    emxEnsureCapacity_real_T1(f_x, i0);
    for (loopUpperBound = 0; loopUpperBound < g_L->size[0]; loopUpperBound++) {
      f_x->data[loopUpperBound] = ((((-g_L->data[loopUpperBound] * (V->
        data[loopUpperBound] - E_L->data[loopUpperBound]) - g_w->
        data[loopUpperBound] * (V->data[loopUpperBound] - E_w->
        data[loopUpperBound])) - g_e->data[loopUpperBound] * (V->
        data[loopUpperBound] - E_e->data[loopUpperBound])) - g_i->
        data[loopUpperBound] * (V->data[loopUpperBound] - E_i->
        data[loopUpperBound])) + I_e->data[loopUpperBound]) + X_t->
        data[loopUpperBound];
    }

    /* External input */
    /* s - Synaptic Output  */
    /* w - Adaptation Variable */
    /* x - Synaptic Trace for STDP */
    i0 = X_t->size[0];
    emxEnsureCapacity_real_T1(X_t, i0);
    for (loopUpperBound = 0; loopUpperBound < X_t->size[0]; loopUpperBound++) {
      X_t->data[loopUpperBound] += dX->data[loopUpperBound];
    }

    i0 = V->size[0];
    emxEnsureCapacity_real_T1(V, i0);
    for (loopUpperBound = 0; loopUpperBound < V->size[0]; loopUpperBound++) {
      V->data[loopUpperBound] += f_x->data[loopUpperBound] / C->
        data[loopUpperBound] * dt;
    }

    i0 = s->size[0];
    emxEnsureCapacity_real_T1(s, i0);
    for (loopUpperBound = 0; loopUpperBound < s->size[0]; loopUpperBound++) {
      s->data[loopUpperBound] += -s->data[loopUpperBound] / tau_s->
        data[loopUpperBound] * dt;
    }

    i0 = w->size[0];
    emxEnsureCapacity_real_T1(w, i0);
    for (loopUpperBound = 0; loopUpperBound < w->size[0]; loopUpperBound++) {
      w->data[loopUpperBound] += (a_w->data[loopUpperBound] * (1.0 - w->
        data[loopUpperBound]) - b_w->data[loopUpperBound] * w->
        data[loopUpperBound]) * dt;
    }

    i0 = x->size[0];
    emxEnsureCapacity_real_T1(x, i0);
    for (loopUpperBound = 0; loopUpperBound < x->size[0]; loopUpperBound++) {
      x->data[loopUpperBound] += -x->data[loopUpperBound] / tauSTDP->
        data[loopUpperBound] * dt;
    }

    /* timecounter = round(timecounter+dt,4);  %Round to deal with computational error */
    /* a_w - Adaptation activation rate for the next time step (unless spike) */
    i0 = g_x->size[0];
    g_x->size[0] = V->size[0];
    emxEnsureCapacity_real_T1(g_x, i0);
    for (loopUpperBound = 0; loopUpperBound < V->size[0]; loopUpperBound++) {
      g_x->data[loopUpperBound] = (V->data[loopUpperBound] - E_L->
        data[loopUpperBound]) * delta_T->data[loopUpperBound];
    }

    nx = g_x->size[0];
    for (ii = 0; ii < nx; ii++) {
      g_x->data[ii] = exp(g_x->data[ii]);
    }

    i0 = a_w->size[0];
    a_w->size[0] = w_r->size[0];
    emxEnsureCapacity_real_T1(a_w, i0);
    for (loopUpperBound = 0; loopUpperBound < w_r->size[0]; loopUpperBound++) {
      a_w->data[loopUpperBound] = w_r->data[loopUpperBound] * b_w->
        data[loopUpperBound] / (1.0 - w_r->data[loopUpperBound]) * g_x->
        data[loopUpperBound];
    }

    /*     %% Spiking */
    i0 = h_x->size[0];
    h_x->size[0] = V->size[0];
    emxEnsureCapacity_boolean_T1(h_x, i0);
    for (loopUpperBound = 0; loopUpperBound < V->size[0]; loopUpperBound++) {
      h_x->data[loopUpperBound] = (V->data[loopUpperBound] > V_th->
        data[loopUpperBound]);
    }

    p = false;
    nm1d2 = 1;
    exitg1 = false;
    while ((!exitg1) && (nm1d2 <= h_x->size[0])) {
      b0 = !h_x->data[nm1d2 - 1];
      if (!b0) {
        p = true;
        exitg1 = true;
      } else {
        nm1d2++;
      }
    }

    if (p) {
      /* Find neurons that crossed threshold and record the spiketimes  */
      i0 = i_x->size[0];
      i_x->size[0] = V->size[0];
      emxEnsureCapacity_boolean_T1(i_x, i0);
      for (loopUpperBound = 0; loopUpperBound < V->size[0]; loopUpperBound++) {
        i_x->data[loopUpperBound] = (V->data[loopUpperBound] > V_th->
          data[loopUpperBound]);
      }

      nx = i_x->size[0];
      idx = 0;
      i0 = c_ii->size[0];
      c_ii->size[0] = i_x->size[0];
      emxEnsureCapacity_int32_T(c_ii, i0);
      nm1d2 = 1;
      exitg1 = false;
      while ((!exitg1) && (nm1d2 <= nx)) {
        if (i_x->data[nm1d2 - 1]) {
          idx++;
          c_ii->data[idx - 1] = nm1d2;
          if (idx >= nx) {
            exitg1 = true;
          } else {
            nm1d2++;
          }
        } else {
          nm1d2++;
        }
      }

      if (i_x->size[0] == 1) {
        if (idx == 0) {
          i0 = c_ii->size[0];
          c_ii->size[0] = 0;
          emxEnsureCapacity_int32_T(c_ii, i0);
        }
      } else {
        i0 = c_ii->size[0];
        if (1 > idx) {
          c_ii->size[0] = 0;
        } else {
          c_ii->size[0] = idx;
        }

        emxEnsureCapacity_int32_T(c_ii, i0);
      }

      if (recordVALs->data[tt] != 0) {
        i0 = c_ii->size[0];
        nm1d2 = r21->size[0];
        r21->size[0] = (int32_T)((real_T)i0 - 1.0) + 1;
        emxEnsureCapacity_int32_T(r21, nm1d2);
        for (loopUpperBound = 0; loopUpperBound <= (int32_T)((real_T)i0 - 1.0);
             loopUpperBound++) {
          r21->data[loopUpperBound] = (int32_T)(cdiff + (1.0 + (real_T)
            loopUpperBound)) - 1;
        }

        i0 = reshapes[0].f1->size[0];
        reshapes[0].f1->size[0] = c_ii->size[0];
        emxEnsureCapacity_real_T1(reshapes[0].f1, i0);
        for (loopUpperBound = 0; loopUpperBound < c_ii->size[0]; loopUpperBound
             ++) {
          reshapes[0].f1->data[loopUpperBound] = -0.0 - dt;
        }

        nm1d2 = reshapes[0].f1->size[0] - 1;
        for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
          spikes->data[r21->data[loopUpperBound]] = (real32_T)reshapes[0]
            .f1->data[loopUpperBound];
        }

        nm1d2 = c_ii->size[0] - 1;
        for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
          spikes->data[r21->data[loopUpperBound] + spikes->size[0]] = (real32_T)
            c_ii->data[loopUpperBound];
        }
      }

      cdiff += (real_T)c_ii->size[0];

      /* Jump the conductance */
      i0 = b_s->size[0];
      b_s->size[0] = c_ii->size[0];
      emxEnsureCapacity_real_T1(b_s, i0);
      for (loopUpperBound = 0; loopUpperBound < c_ii->size[0]; loopUpperBound++)
      {
        b_s->data[loopUpperBound] = s->data[c_ii->data[loopUpperBound] - 1] +
          1.0;
      }

      for (loopUpperBound = 0; loopUpperBound < b_s->size[0]; loopUpperBound++)
      {
        s->data[c_ii->data[loopUpperBound] - 1] = b_s->data[loopUpperBound];
      }

      /* Set spiking neurons refractory period  */
      for (loopUpperBound = 0; loopUpperBound < c_ii->size[0]; loopUpperBound++)
      {
        t_r->data[c_ii->data[loopUpperBound] - 1] = t_ref->data[c_ii->
          data[loopUpperBound] - 1];
      }

      /* Jump the synaptic trace */
      i0 = k_x->size[0];
      k_x->size[0] = c_ii->size[0];
      emxEnsureCapacity_real_T1(k_x, i0);
      for (loopUpperBound = 0; loopUpperBound < c_ii->size[0]; loopUpperBound++)
      {
        k_x->data[loopUpperBound] = x->data[c_ii->data[loopUpperBound] - 1] +
          1.0;
      }

      for (loopUpperBound = 0; loopUpperBound < k_x->size[0]; loopUpperBound++)
      {
        x->data[c_ii->data[loopUpperBound] - 1] = k_x->data[loopUpperBound];
      }

      /* Implement STDP (Vogels 2011 SuppEqn 4/5) I->E only */
      /* Presynaptic I Cells */
      /* PreIspikes = intersect(spikeneurons,Icells); */
      n = c_ii->size[0] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (c_ii->data[mA] > EPopNum) {
          nm1d2++;
        }
      }

      i0 = PreIspikes->size[0];
      PreIspikes->size[0] = nm1d2;
      emxEnsureCapacity_real_T1(PreIspikes, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (c_ii->data[mA] > EPopNum) {
          PreIspikes->data[nm1d2] = c_ii->data[mA];
          nm1d2++;
        }
      }

      n = EcellIDX->size[1] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EcellIDX->data[mA]) {
          nm1d2++;
        }
      }

      i0 = r22->size[0] * r22->size[1];
      r22->size[0] = 1;
      r22->size[1] = nm1d2;
      emxEnsureCapacity_int32_T1(r22, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EcellIDX->data[mA]) {
          r22->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      i0 = r19->size[0];
      r19->size[0] = ELearningRate->size[0];
      emxEnsureCapacity_real_T1(r19, i0);
      for (loopUpperBound = 0; loopUpperBound < ELearningRate->size[0];
           loopUpperBound++) {
        r19->data[loopUpperBound] = ELearningRate->data[loopUpperBound] *
          (x->data[r22->data[r22->size[0] * loopUpperBound] - 1] - alpha->
           data[loopUpperBound]);
      }

      n = EcellIDX->size[1] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EcellIDX->data[mA]) {
          nm1d2++;
        }
      }

      i0 = r23->size[0] * r23->size[1];
      r23->size[0] = 1;
      r23->size[1] = nm1d2;
      emxEnsureCapacity_int32_T1(r23, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EcellIDX->data[mA]) {
          r23->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      ndbl = r19->data[0];
      i0 = b_EI_mat->size[0] * b_EI_mat->size[1];
      b_EI_mat->size[0] = r23->size[1];
      b_EI_mat->size[1] = PreIspikes->size[0];
      emxEnsureCapacity_real_T(b_EI_mat, i0);
      for (i0 = 0; i0 < PreIspikes->size[0]; i0++) {
        for (loopUpperBound = 0; loopUpperBound < r23->size[1]; loopUpperBound++)
        {
          b_EI_mat->data[loopUpperBound + b_EI_mat->size[0] * i0] = EI_mat->
            data[(r23->data[r23->size[0] * loopUpperBound] + EI_mat->size[0] *
                  ((int32_T)PreIspikes->data[i0] - 1)) - 1] + ndbl;
        }
      }

      for (i0 = 0; i0 < b_EI_mat->size[1]; i0++) {
        for (loopUpperBound = 0; loopUpperBound < b_EI_mat->size[0];
             loopUpperBound++) {
          EI_mat->data[(r23->data[r23->size[0] * loopUpperBound] + EI_mat->size
                        [0] * ((int32_T)PreIspikes->data[i0] - 1)) - 1] =
            b_EI_mat->data[loopUpperBound + b_EI_mat->size[0] * i0];
        }
      }

      /* Postsynaptic E cells */
      /* PostEspikes = intersect(spikeneurons,Ecells); */
      n = c_ii->size[0] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (c_ii->data[mA] <= EPopNum) {
          nm1d2++;
        }
      }

      i0 = PostEspikes->size[0];
      PostEspikes->size[0] = nm1d2;
      emxEnsureCapacity_real_T1(PostEspikes, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (c_ii->data[mA] <= EPopNum) {
          PostEspikes->data[nm1d2] = c_ii->data[mA];
          nm1d2++;
        }
      }

      n = IcellIDX->size[1] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (IcellIDX->data[mA]) {
          nm1d2++;
        }
      }

      i0 = r24->size[0] * r24->size[1];
      r24->size[0] = 1;
      r24->size[1] = nm1d2;
      emxEnsureCapacity_int32_T1(r24, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (IcellIDX->data[mA]) {
          r24->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      i0 = r20->size[0] * r20->size[1];
      r20->size[0] = 1;
      r20->size[1] = ILearningRate->size[1];
      emxEnsureCapacity_real_T(r20, i0);
      for (loopUpperBound = 0; loopUpperBound < ILearningRate->size[0] *
           ILearningRate->size[1]; loopUpperBound++) {
        r20->data[loopUpperBound] = ILearningRate->data[loopUpperBound] *
          x->data[r24->data[loopUpperBound] - 1];
      }

      n = IcellIDX->size[1] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (IcellIDX->data[mA]) {
          nm1d2++;
        }
      }

      i0 = r25->size[0] * r25->size[1];
      r25->size[0] = 1;
      r25->size[1] = nm1d2;
      emxEnsureCapacity_int32_T1(r25, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (IcellIDX->data[mA]) {
          r25->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      ndbl = r20->data[0];
      i0 = c_EI_mat->size[0] * c_EI_mat->size[1];
      c_EI_mat->size[0] = PostEspikes->size[0];
      c_EI_mat->size[1] = r25->size[1];
      emxEnsureCapacity_real_T(c_EI_mat, i0);
      for (i0 = 0; i0 < r25->size[1]; i0++) {
        for (loopUpperBound = 0; loopUpperBound < PostEspikes->size[0];
             loopUpperBound++) {
          c_EI_mat->data[loopUpperBound + c_EI_mat->size[0] * i0] = EI_mat->
            data[((int32_T)PostEspikes->data[loopUpperBound] + EI_mat->size[0] *
                  (r25->data[r25->size[0] * i0] - 1)) - 1] + ndbl;
        }
      }

      for (i0 = 0; i0 < c_EI_mat->size[1]; i0++) {
        for (loopUpperBound = 0; loopUpperBound < c_EI_mat->size[0];
             loopUpperBound++) {
          EI_mat->data[((int32_T)PostEspikes->data[loopUpperBound] +
                        EI_mat->size[0] * (r25->data[r25->size[0] * i0] - 1)) -
            1] = c_EI_mat->data[loopUpperBound + c_EI_mat->size[0] * i0];
        }
      }

      i0 = EI_mat->size[0] * EI_mat->size[1];
      nm1d2 = EI_mat->size[0] * EI_mat->size[1];
      emxEnsureCapacity_real_T(EI_mat, nm1d2);
      for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
        EI_mat->data[loopUpperBound] *= (real_T)isconnected->data[loopUpperBound];
      }

      /* Keep only connected pairs */
      n = EI_mat->size[0] * EI_mat->size[1] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EI_mat->data[mA] <= 0.0) {
          nm1d2++;
        }
      }

      i0 = r26->size[0];
      r26->size[0] = nm1d2;
      emxEnsureCapacity_int32_T(r26, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (EI_mat->data[mA] <= 0.0) {
          r26->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      i0 = r26->size[0];
      for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
        EI_mat->data[r26->data[loopUpperBound] - 1] = 0.0;
      }

      /* Get rid of any negative synapses... */
    }

    /*     %%  Refractory period Countdowns */
    i0 = j_x->size[0];
    j_x->size[0] = t_r->size[0];
    emxEnsureCapacity_boolean_T1(j_x, i0);
    for (loopUpperBound = 0; loopUpperBound < t_r->size[0]; loopUpperBound++) {
      j_x->data[loopUpperBound] = (t_r->data[loopUpperBound] > 0.0);
    }

    p = false;
    nm1d2 = 1;
    exitg1 = false;
    while ((!exitg1) && (nm1d2 <= j_x->size[0])) {
      b0 = !j_x->data[nm1d2 - 1];
      if (!b0) {
        p = true;
        exitg1 = true;
      } else {
        nm1d2++;
      }
    }

    if (p) {
      /* Hold voltage, synaptic/adaptation rates at spike levels */
      n = t_r->size[0] - 1;
      for (ii = 0; ii <= n; ii++) {
        if (t_r->data[ii] > 0.0) {
          V->data[ii] = V_reset->data[ii];
        }
      }

      n = t_r->size[0] - 1;
      for (ii = 0; ii <= n; ii++) {
        if (t_r->data[ii] > 0.0) {
          a_w->data[ii] = b->data[ii];
        }
      }

      /* Count down the refractory period */
      n = t_r->size[0] - 1;
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (t_r->data[mA] > 0.0) {
          nm1d2++;
        }
      }

      i0 = r27->size[0];
      r27->size[0] = nm1d2;
      emxEnsureCapacity_int32_T(r27, i0);
      nm1d2 = 0;
      for (mA = 0; mA <= n; mA++) {
        if (t_r->data[mA] > 0.0) {
          r27->data[nm1d2] = mA + 1;
          nm1d2++;
        }
      }

      i0 = b_t_r->size[0];
      b_t_r->size[0] = r27->size[0];
      emxEnsureCapacity_real_T1(b_t_r, i0);
      for (loopUpperBound = 0; loopUpperBound < r27->size[0]; loopUpperBound++)
      {
        b_t_r->data[loopUpperBound] = t_r->data[r27->data[loopUpperBound] - 1] -
          dt;
      }

      for (loopUpperBound = 0; loopUpperBound < b_t_r->size[0]; loopUpperBound++)
      {
        t_r->data[r27->data[loopUpperBound] - 1] = b_t_r->data[loopUpperBound];
      }
    }

    /*     %% Synaptic,Adaptaion Conductances for the next time step */
    i0 = g_w->size[0];
    g_w->size[0] = gwnorm->size[0];
    emxEnsureCapacity_real_T1(g_w, i0);
    for (loopUpperBound = 0; loopUpperBound < gwnorm->size[0]; loopUpperBound++)
    {
      g_w->data[loopUpperBound] = gwnorm->data[loopUpperBound] * w->
        data[loopUpperBound];
    }

    if ((EE_mat->size[1] == 1) || (s->size[0] == 1)) {
      i0 = b_C->size[0];
      b_C->size[0] = EE_mat->size[0];
      emxEnsureCapacity_real_T1(b_C, i0);
      nm1d2 = EE_mat->size[0] - 1;
      for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
        b_C->data[loopUpperBound] = 0.0;
        loop_ub = EE_mat->size[1] - 1;
        for (ii = 0; ii <= loop_ub; ii++) {
          b_C->data[loopUpperBound] += EE_mat->data[loopUpperBound +
            EE_mat->size[0] * ii] * s->data[ii];
        }
      }
    } else {
      mA = EE_mat->size[0];
      nA = EE_mat->size[1];
      if ((EE_mat->size[0] == 0) || (EE_mat->size[1] == 0) || (s->size[0] == 0))
      {
        i0 = b_C->size[0];
        b_C->size[0] = EE_mat->size[0];
        emxEnsureCapacity_real_T1(b_C, i0);
        for (loopUpperBound = 0; loopUpperBound < EE_mat->size[0];
             loopUpperBound++) {
          b_C->data[loopUpperBound] = 0.0;
        }
      } else {
        i0 = b_C->size[0];
        b_C->size[0] = EE_mat->size[0];
        emxEnsureCapacity_real_T1(b_C, i0);
        for (loopUpperBound = 0; loopUpperBound < EE_mat->size[0];
             loopUpperBound++) {
          b_C->data[loopUpperBound] = 0.0;
        }

        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          for (ii = 0; ii < mA; ii++) {
            b_C->data[ii] = 0.0;
          }

          nm1d2 = mA;
        }

        br = 0;
        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          i0 = br + nA;
          for (n = 0; n < i0 - br; n++) {
            idx = br + n;
            if (s->data[idx] != 0.0) {
              nx = nm1d2;
              for (k = 0; k < mA; k++) {
                nx++;
                b_C->data[k] += s->data[idx] * EE_mat->data[nx - 1];
              }
            }

            nm1d2 += mA;
          }

          br += nA;
          nm1d2 = mA;
        }
      }
    }

    if ((IE_mat->size[1] == 1) || (s->size[0] == 1)) {
      i0 = c_C->size[0];
      c_C->size[0] = IE_mat->size[0];
      emxEnsureCapacity_real_T1(c_C, i0);
      nm1d2 = IE_mat->size[0] - 1;
      for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
        c_C->data[loopUpperBound] = 0.0;
        loop_ub = IE_mat->size[1] - 1;
        for (ii = 0; ii <= loop_ub; ii++) {
          c_C->data[loopUpperBound] += IE_mat->data[loopUpperBound +
            IE_mat->size[0] * ii] * s->data[ii];
        }
      }
    } else {
      mA = IE_mat->size[0];
      nA = IE_mat->size[1];
      if ((IE_mat->size[0] == 0) || (IE_mat->size[1] == 0) || (s->size[0] == 0))
      {
        i0 = c_C->size[0];
        c_C->size[0] = IE_mat->size[0];
        emxEnsureCapacity_real_T1(c_C, i0);
        for (loopUpperBound = 0; loopUpperBound < IE_mat->size[0];
             loopUpperBound++) {
          c_C->data[loopUpperBound] = 0.0;
        }
      } else {
        i0 = c_C->size[0];
        c_C->size[0] = IE_mat->size[0];
        emxEnsureCapacity_real_T1(c_C, i0);
        for (loopUpperBound = 0; loopUpperBound < IE_mat->size[0];
             loopUpperBound++) {
          c_C->data[loopUpperBound] = 0.0;
        }

        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          for (ii = 0; ii < mA; ii++) {
            c_C->data[ii] = 0.0;
          }

          nm1d2 = mA;
        }

        br = 0;
        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          i0 = br + nA;
          for (n = 0; n < i0 - br; n++) {
            idx = br + n;
            if (s->data[idx] != 0.0) {
              nx = nm1d2;
              for (k = 0; k < mA; k++) {
                nx++;
                c_C->data[k] += s->data[idx] * IE_mat->data[nx - 1];
              }
            }

            nm1d2 += mA;
          }

          br += nA;
          nm1d2 = mA;
        }
      }
    }

    i0 = g_e->size[0];
    g_e->size[0] = b_C->size[0];
    emxEnsureCapacity_real_T1(g_e, i0);
    for (loopUpperBound = 0; loopUpperBound < b_C->size[0]; loopUpperBound++) {
      g_e->data[loopUpperBound] = b_C->data[loopUpperBound] + c_C->
        data[loopUpperBound];
    }

    if ((II_mat->size[1] == 1) || (s->size[0] == 1)) {
      i0 = d_C->size[0];
      d_C->size[0] = II_mat->size[0];
      emxEnsureCapacity_real_T1(d_C, i0);
      nm1d2 = II_mat->size[0] - 1;
      for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
        d_C->data[loopUpperBound] = 0.0;
        loop_ub = II_mat->size[1] - 1;
        for (ii = 0; ii <= loop_ub; ii++) {
          d_C->data[loopUpperBound] += II_mat->data[loopUpperBound +
            II_mat->size[0] * ii] * s->data[ii];
        }
      }
    } else {
      mA = II_mat->size[0];
      nA = II_mat->size[1];
      if ((II_mat->size[0] == 0) || (II_mat->size[1] == 0) || (s->size[0] == 0))
      {
        i0 = d_C->size[0];
        d_C->size[0] = II_mat->size[0];
        emxEnsureCapacity_real_T1(d_C, i0);
        for (loopUpperBound = 0; loopUpperBound < II_mat->size[0];
             loopUpperBound++) {
          d_C->data[loopUpperBound] = 0.0;
        }
      } else {
        i0 = d_C->size[0];
        d_C->size[0] = II_mat->size[0];
        emxEnsureCapacity_real_T1(d_C, i0);
        for (loopUpperBound = 0; loopUpperBound < II_mat->size[0];
             loopUpperBound++) {
          d_C->data[loopUpperBound] = 0.0;
        }

        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          for (ii = 0; ii < mA; ii++) {
            d_C->data[ii] = 0.0;
          }

          nm1d2 = mA;
        }

        br = 0;
        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          i0 = br + nA;
          for (n = 0; n < i0 - br; n++) {
            idx = br + n;
            if (s->data[idx] != 0.0) {
              nx = nm1d2;
              for (k = 0; k < mA; k++) {
                nx++;
                d_C->data[k] += s->data[idx] * II_mat->data[nx - 1];
              }
            }

            nm1d2 += mA;
          }

          br += nA;
          nm1d2 = mA;
        }
      }
    }

    if ((EI_mat->size[1] == 1) || (s->size[0] == 1)) {
      i0 = e_C->size[0];
      e_C->size[0] = EI_mat->size[0];
      emxEnsureCapacity_real_T1(e_C, i0);
      nm1d2 = EI_mat->size[0] - 1;
      for (loopUpperBound = 0; loopUpperBound <= nm1d2; loopUpperBound++) {
        e_C->data[loopUpperBound] = 0.0;
        loop_ub = EI_mat->size[1] - 1;
        for (ii = 0; ii <= loop_ub; ii++) {
          e_C->data[loopUpperBound] += EI_mat->data[loopUpperBound +
            EI_mat->size[0] * ii] * s->data[ii];
        }
      }
    } else {
      mA = EI_mat->size[0];
      nA = EI_mat->size[1];
      if ((EI_mat->size[0] == 0) || (EI_mat->size[1] == 0) || (s->size[0] == 0))
      {
        i0 = e_C->size[0];
        e_C->size[0] = EI_mat->size[0];
        emxEnsureCapacity_real_T1(e_C, i0);
        for (loopUpperBound = 0; loopUpperBound < EI_mat->size[0];
             loopUpperBound++) {
          e_C->data[loopUpperBound] = 0.0;
        }
      } else {
        i0 = e_C->size[0];
        e_C->size[0] = EI_mat->size[0];
        emxEnsureCapacity_real_T1(e_C, i0);
        for (loopUpperBound = 0; loopUpperBound < EI_mat->size[0];
             loopUpperBound++) {
          e_C->data[loopUpperBound] = 0.0;
        }

        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          for (ii = 0; ii < mA; ii++) {
            e_C->data[ii] = 0.0;
          }

          nm1d2 = mA;
        }

        br = 0;
        nm1d2 = 0;
        while ((mA > 0) && (nm1d2 <= 0)) {
          i0 = br + nA;
          for (n = 0; n < i0 - br; n++) {
            idx = br + n;
            if (s->data[idx] != 0.0) {
              nx = nm1d2;
              for (k = 0; k < mA; k++) {
                nx++;
                e_C->data[k] += s->data[idx] * EI_mat->data[nx - 1];
              }
            }

            nm1d2 += mA;
          }

          br += nA;
          nm1d2 = mA;
        }
      }
    }

    i0 = g_i->size[0];
    g_i->size[0] = d_C->size[0];
    emxEnsureCapacity_real_T1(g_i, i0);
    for (loopUpperBound = 0; loopUpperBound < d_C->size[0]; loopUpperBound++) {
      g_i->data[loopUpperBound] = d_C->data[loopUpperBound] + e_C->
        data[loopUpperBound];
    }

    /*     %% Add data to the output variables */
    /* Question: is accessing structure slower than doubles? */
    if ((f_r == 0.0) && (-0.0 - dt >= 0.0) && (recordVALs->data[tt] != 0)) {
      SimValues->t->data[(int32_T)savecounter - 1] = -0.0 - dt;
      for (i0 = 0; i0 < V->size[0]; i0++) {
        SimValues->V->data[i0 + SimValues->V->size[0] * ((int32_T)savecounter -
          1)] = V->data[i0];
      }

      for (i0 = 0; i0 < g_w->size[0]; i0++) {
        SimValues->g_w->data[i0 + SimValues->g_w->size[0] * ((int32_T)
          savecounter - 1)] = g_w->data[i0];
      }

      for (i0 = 0; i0 < g_e->size[0]; i0++) {
        SimValues->g_e->data[i0 + SimValues->g_e->size[0] * ((int32_T)
          savecounter - 1)] = g_e->data[i0];
      }

      for (i0 = 0; i0 < g_i->size[0]; i0++) {
        SimValues->g_i->data[i0 + SimValues->g_i->size[0] * ((int32_T)
          savecounter - 1)] = g_i->data[i0];
      }

      for (i0 = 0; i0 < s->size[0]; i0++) {
        SimValues->s->data[i0 + SimValues->s->size[0] * ((int32_T)savecounter -
          1)] = s->data[i0];
      }

      for (i0 = 0; i0 < w->size[0]; i0++) {
        SimValues->w->data[i0 + SimValues->w->size[0] * ((int32_T)savecounter -
          1)] = w->data[i0];
      }

      for (i0 = 0; i0 < a_w->size[0]; i0++) {
        SimValues->a_w->data[i0 + SimValues->a_w->size[0] * ((int32_T)
          savecounter - 1)] = a_w->data[i0];
      }

      for (i0 = 0; i0 < I_e->size[0]; i0++) {
        SimValues->Input->data[i0 + SimValues->Input->size[0] * ((int32_T)
          savecounter - 1)] = I_e->data[i0] + X_t->data[i0];
      }

      savecounter++;
    }

    /* Save the weights (less frequently for space concerns) */
    if ((g_r == 0.0) && (-0.0 - dt >= 0.0) && (recordVALs->data[tt] != 0)) {
      SimValues->t_weight->data[(int32_T)weightcounter - 1] = -0.0 - dt;
      for (i0 = 0; i0 < EE_mat->size[1]; i0++) {
        for (nm1d2 = 0; nm1d2 < EE_mat->size[0]; nm1d2++) {
          SimValues->WeightMat->data[(nm1d2 + SimValues->WeightMat->size[0] * i0)
            + SimValues->WeightMat->size[0] * SimValues->WeightMat->size[1] *
            ((int32_T)weightcounter - 1)] = ((EE_mat->data[nm1d2 + EE_mat->size
            [0] * i0] + II_mat->data[nm1d2 + II_mat->size[0] * i0]) +
            EI_mat->data[nm1d2 + EI_mat->size[0] * i0]) + IE_mat->data[nm1d2 +
            IE_mat->size[0] * i0];
        }
      }

      weightcounter++;
    }

    /* Idea: add a catch for silent network or excessive firing network? */
  }

  emxFree_real_T(&b_t_r);
  emxFree_real_T(&c_EI_mat);
  emxFree_real_T(&b_EI_mat);
  emxFree_real_T(&k_x);
  emxFree_real_T(&b_s);
  emxFree_real_T(&e_C);
  emxFree_real_T(&d_C);
  emxFree_real_T(&c_C);
  emxFree_real_T(&b_C);
  emxFree_boolean_T(&j_x);
  emxFreeMatrix_cell_wrap_1(reshapes);
  emxFree_int32_T(&c_ii);
  emxFree_boolean_T(&i_x);
  emxFree_boolean_T(&h_x);
  emxFree_real_T(&g_x);
  emxFree_real_T(&f_x);
  emxFree_real_T(&e_x);
  emxFree_boolean_T(&d_x);
  emxFree_int32_T(&r27);
  emxFree_int32_T(&r26);
  emxFree_int32_T(&r25);
  emxFree_int32_T(&r24);
  emxFree_int32_T(&r23);
  emxFree_int32_T(&r22);
  emxFree_int32_T(&r21);
  emxFree_real_T(&r20);
  emxFree_real_T(&r19);
  emxFree_real_T(&r18);
  emxFree_real_T(&PostEspikes);
  emxFree_real_T(&PreIspikes);
  emxFree_real_T(&dX);
  emxFree_real_T(&x);
  emxFree_real_T(&t_r);
  emxFree_real_T(&X_t);
  emxFree_real_T(&w);
  emxFree_real_T(&s);
  emxFree_real_T(&a_w);
  emxFree_real_T(&g_w);
  emxFree_real_T(&g_i);
  emxFree_real_T(&g_e);
  emxFree_real_T(&V);
  emxFree_real_T(&alpha);
  emxFree_real_T(&tauSTDP);
  emxFree_real_T(&ILearningRate);
  emxFree_real_T(&ELearningRate);
  emxFree_real_T(&tau_s);
  emxFree_real_T(&E_i);
  emxFree_real_T(&E_e);
  emxFree_real_T(&b);
  emxFree_real_T(&w_r);
  emxFree_real_T(&gwnorm);
  emxFree_real_T(&delta_T);
  emxFree_real_T(&b_w);
  emxFree_real_T(&E_w);
  emxFree_real_T(&theta);
  emxFree_real_T(&sigma);
  emxFree_real_T(&t_ref);
  emxFree_real_T(&V_reset);
  emxFree_real_T(&V_th);
  emxFree_real_T(&I_e);
  emxFree_real_T(&C);
  emxFree_real_T(&g_L);
  emxFree_real_T(&E_L);
  emxFree_boolean_T(&isconnected);
  emxFree_boolean_T(&IcellIDX);
  emxFree_boolean_T(&EcellIDX);
  emxFree_real_T(&EI_mat);
  emxFree_real_T(&IE_mat);
  emxFree_real_T(&II_mat);
  emxFree_real_T(&EE_mat);
  emxFree_int8_T(&recordVALs);
  emxInit_int32_T1(&b_idx, 2, true);

  /*  */
  /* Catch for no spiking in simulation error */
  ndbl = cdiff + 1.0;
  i0 = spikes->size[0];
  nm1d2 = b_idx->size[0] * b_idx->size[1];
  b_idx->size[0] = 1;
  b_idx->size[1] = (int32_T)((real_T)i0 - (cdiff + 1.0)) + 1;
  emxEnsureCapacity_int32_T1(b_idx, nm1d2);
  for (loopUpperBound = 0; loopUpperBound <= (int32_T)((real_T)i0 - ndbl);
       loopUpperBound++) {
    b_idx->data[b_idx->size[0] * loopUpperBound] = (int32_T)((cdiff + 1.0) +
      (real_T)loopUpperBound);
  }

  br = spikes->size[0] - 1;
  if (b_idx->size[1] == 1) {
    idx = spikes->size[0] - 1;
    for (nx = 0; nx < 2; nx++) {
      nm1d2 = b_idx->data[0];
      for (mA = 0; mA <= br - nm1d2; mA++) {
        n = nm1d2 + mA;
        spikes->data[(n + spikes->size[0] * nx) - 1] = spikes->data[n +
          spikes->size[0] * nx];
      }
    }
  } else {
    emxInit_boolean_T(&b_b, 2, true);
    i0 = b_b->size[0] * b_b->size[1];
    b_b->size[0] = 1;
    b_b->size[1] = spikes->size[0];
    emxEnsureCapacity_boolean_T(b_b, i0);
    for (loopUpperBound = 0; loopUpperBound < spikes->size[0]; loopUpperBound++)
    {
      b_b->data[loopUpperBound] = false;
    }

    b_loopUpperBound = b_idx->size[1];
    for (ii = 0; ii < b_loopUpperBound; ii++) {
      b_b->data[b_idx->data[ii] - 1] = true;
    }

    n = 0;
    b_loopUpperBound = b_b->size[1];
    for (k = 0; k < b_loopUpperBound; k++) {
      n += b_b->data[k];
    }

    idx = spikes->size[0] - n;
    mA = 0;
    for (k = 0; k <= br; k++) {
      if ((1 + k > b_b->size[1]) || (!b_b->data[k])) {
        for (nx = 0; nx < 2; nx++) {
          spikes->data[mA + spikes->size[0] * nx] = spikes->data[k +
            spikes->size[0] * nx];
        }

        mA++;
      }
    }

    emxFree_boolean_T(&b_b);
  }

  emxFree_int32_T(&b_idx);
  if (1 > idx) {
    i0 = 0;
  } else {
    i0 = idx;
  }

  emxInit_real32_T(&b_spikes, 2, true);
  nm1d2 = b_spikes->size[0] * b_spikes->size[1];
  b_spikes->size[0] = i0;
  b_spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(b_spikes, nm1d2);
  for (nm1d2 = 0; nm1d2 < 2; nm1d2++) {
    for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
      b_spikes->data[loopUpperBound + b_spikes->size[0] * nm1d2] = spikes->
        data[loopUpperBound + spikes->size[0] * nm1d2];
    }
  }

  i0 = spikes->size[0] * spikes->size[1];
  spikes->size[0] = b_spikes->size[0];
  spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(spikes, i0);
  for (i0 = 0; i0 < 2; i0++) {
    for (loopUpperBound = 0; loopUpperBound < b_spikes->size[0]; loopUpperBound
         ++) {
      spikes->data[loopUpperBound + spikes->size[0] * i0] = b_spikes->
        data[loopUpperBound + b_spikes->size[0] * i0];
    }
  }

  emxFree_real32_T(&b_spikes);

  /* if isempty(spikes); spikes = [nan nan]; end */
  /*  Figure */
  emlrtRandu(&ndbl, 1);
  i0 = spikes->size[0];
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    emxInit_boolean_T1(&r28, 1, true);
    emxFree_boolean_T(&r28);
  }

  emxInit_real32_T1(&u, 1, true);
  figure(&emlrtMCI);
  i0 = spikes->size[0];
  nm1d2 = u->size[0];
  u->size[0] = i0;
  emxEnsureCapacity_real32_T1(u, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    u->data[loopUpperBound] = spikes->data[loopUpperBound];
  }

  c_y = NULL;
  m0 = emlrtCreateNumericArray(1, *(int32_T (*)[1])u->size, mxSINGLE_CLASS,
    mxREAL);
  pData = (real32_T *)emlrtMxGetData(m0);
  i0 = 0;
  b_loopUpperBound = spikes->size[0];
  emxFree_real32_T(&u);
  for (mA = 0; mA < b_loopUpperBound; mA++) {
    pData[i0] = spikes->data[mA];
    i0++;
  }

  emxInit_real32_T1(&b_u, 1, true);
  emlrtAssign(&c_y, m0);
  i0 = spikes->size[0];
  nm1d2 = b_u->size[0];
  b_u->size[0] = i0;
  emxEnsureCapacity_real32_T1(b_u, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    b_u->data[loopUpperBound] = spikes->data[loopUpperBound + spikes->size[0]];
  }

  d_y = NULL;
  m0 = emlrtCreateNumericArray(1, *(int32_T (*)[1])b_u->size, mxSINGLE_CLASS,
    mxREAL);
  pData = (real32_T *)emlrtMxGetData(m0);
  i0 = 0;
  b_loopUpperBound = spikes->size[0];
  emxFree_real32_T(&b_u);
  for (mA = 0; mA < b_loopUpperBound; mA++) {
    pData[i0] = spikes->data[mA + spikes->size[0]];
    i0++;
  }

  emlrtAssign(&d_y, m0);
  e_y = NULL;
  m0 = emlrtCreateCharArray(2, iv1);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 2, m0, &c_u[0]);
  emlrtAssign(&e_y, m0);
  f_y = NULL;
  m0 = emlrtCreateCharArray(2, iv2);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 10, m0, &d_u[0]);
  emlrtAssign(&f_y, m0);
  g_y = NULL;
  m0 = emlrtCreateDoubleScalar(0.1);
  emlrtAssign(&g_y, m0);
  plot(c_y, d_y, e_y, f_y, g_y, &b_emlrtMCI);
  c_y = NULL;
  m0 = emlrtCreateCharArray(2, iv3);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 2, m0, &e_u[0]);
  emlrtAssign(&c_y, m0);
  hold(c_y, &c_emlrtMCI);
  c_y = NULL;
  m0 = emlrtCreateNumericArray(2, iv4, mxDOUBLE_CLASS, mxREAL);
  b_pData = emlrtMxGetPr(m0);
  for (mA = 0; mA < 2; mA++) {
    b_pData[mA] = 0.0;
  }

  emlrtAssign(&c_y, m0);
  V0range[0] = 0.0;
  V0range[1] = PopNum;
  d_y = NULL;
  m0 = emlrtCreateNumericArray(2, iv5, mxDOUBLE_CLASS, mxREAL);
  b_pData = emlrtMxGetPr(m0);
  for (mA = 0; mA < 2; mA++) {
    b_pData[mA] = V0range[mA];
  }

  emlrtAssign(&d_y, m0);
  e_y = NULL;
  m0 = emlrtCreateString1('r');
  emlrtAssign(&e_y, m0);
  b_plot(c_y, d_y, e_y, &d_emlrtMCI);
  c_y = NULL;
  m0 = emlrtCreateCharArray(2, iv6);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 9, m0, &f_u[0]);
  emlrtAssign(&c_y, m0);
  xlabel(c_y, &e_emlrtMCI);
  c_y = NULL;
  m0 = emlrtCreateCharArray(2, iv7);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 9, m0, &g_u[0]);
  emlrtAssign(&c_y, m0);
  ylabel(c_y, &f_emlrtMCI);
  c_y = NULL;
  m0 = emlrtCreateCharArray(2, iv8);
  emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 11, m0, &h_u[0]);
  emlrtAssign(&c_y, m0);
  title(c_y, &g_emlrtMCI);
  V0range[0] = -0.0;
  V0range[1] = TimeParams->SimTime;
  c_y = NULL;
  m0 = emlrtCreateNumericArray(2, iv9, mxDOUBLE_CLASS, mxREAL);
  b_pData = emlrtMxGetPr(m0);
  for (mA = 0; mA < 2; mA++) {
    b_pData[mA] = V0range[mA];
  }

  emlrtAssign(&c_y, m0);
  xlim(c_y, &h_emlrtMCI);
  V0range[0] = 0.0;
  V0range[1] = PopNum + 1.0;
  c_y = NULL;
  m0 = emlrtCreateNumericArray(2, iv10, mxDOUBLE_CLASS, mxREAL);
  b_pData = emlrtMxGetPr(m0);
  for (mA = 0; mA < 2; mA++) {
    b_pData[mA] = V0range[mA];
  }

  emxInit_boolean_T1(&c_idx, 1, true);
  emlrtAssign(&c_y, m0);
  ylim(c_y, &i_emlrtMCI);

  /*  Output Structure */
  /* Remove onset time */
  i0 = spikes->size[0];
  nm1d2 = c_idx->size[0];
  c_idx->size[0] = i0;
  emxEnsureCapacity_boolean_T1(c_idx, nm1d2);
  for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
    c_idx->data[loopUpperBound] = (spikes->data[loopUpperBound] <= 0.0F);
  }

  br = spikes->size[0];
  n = 0;
  b_loopUpperBound = c_idx->size[0];
  for (k = 0; k < b_loopUpperBound; k++) {
    n += c_idx->data[k];
  }

  idx = spikes->size[0] - n;
  mA = 0;
  for (k = 0; k < br; k++) {
    if ((1 + k > c_idx->size[0]) || (!c_idx->data[k])) {
      for (nx = 0; nx < 2; nx++) {
        spikes->data[mA + spikes->size[0] * nx] = spikes->data[k + spikes->size
          [0] * nx];
      }

      mA++;
    }
  }

  emxFree_boolean_T(&c_idx);
  if (1 > idx) {
    i0 = 0;
  } else {
    i0 = idx;
  }

  emxInit_real32_T(&c_spikes, 2, true);
  nm1d2 = c_spikes->size[0] * c_spikes->size[1];
  c_spikes->size[0] = i0;
  c_spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(c_spikes, nm1d2);
  for (nm1d2 = 0; nm1d2 < 2; nm1d2++) {
    for (loopUpperBound = 0; loopUpperBound < i0; loopUpperBound++) {
      c_spikes->data[loopUpperBound + c_spikes->size[0] * nm1d2] = spikes->
        data[loopUpperBound + spikes->size[0] * nm1d2];
    }
  }

  i0 = spikes->size[0] * spikes->size[1];
  spikes->size[0] = c_spikes->size[0];
  spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(spikes, i0);
  for (i0 = 0; i0 < 2; i0++) {
    for (loopUpperBound = 0; loopUpperBound < c_spikes->size[0]; loopUpperBound
         ++) {
      spikes->data[loopUpperBound + spikes->size[0] * i0] = c_spikes->
        data[loopUpperBound + c_spikes->size[0] * i0];
    }
  }

  emxFree_real32_T(&c_spikes);

  /* if cellout */
  /*     for cc = 1:PopNum %This can go very slow with lots of spikes.... */
  /*         spikesbycell{cc} = spikes(spikes(:,2)==cc,1); */
  /*     end */
  /*     SimValues.spikesbycell    = spikesbycell; */
  /* end */
  i0 = SimValues->spikes->size[0] * SimValues->spikes->size[1];
  SimValues->spikes->size[0] = spikes->size[0];
  SimValues->spikes->size[1] = 2;
  emxEnsureCapacity_real32_T(SimValues->spikes, i0);
  for (i0 = 0; i0 < spikes->size[0] * spikes->size[1]; i0++) {
    SimValues->spikes->data[i0] = spikes->data[i0];
  }

  emxFree_real32_T(&spikes);
  i0 = SimValues->EcellIDX->size[0] * SimValues->EcellIDX->size[1];
  SimValues->EcellIDX->size[0] = 1;
  SimValues->EcellIDX->size[1] = Ecells->size[1];
  emxEnsureCapacity_real_T(SimValues->EcellIDX, i0);
  for (i0 = 0; i0 < Ecells->size[0] * Ecells->size[1]; i0++) {
    SimValues->EcellIDX->data[i0] = Ecells->data[i0];
  }

  emxFree_real_T(&Ecells);
  i0 = SimValues->IcellIDX->size[0] * SimValues->IcellIDX->size[1];
  SimValues->IcellIDX->size[0] = 1;
  SimValues->IcellIDX->size[1] = Icells->size[1];
  emxEnsureCapacity_real_T(SimValues->IcellIDX, i0);
  for (i0 = 0; i0 < Icells->size[0] * Icells->size[1]; i0++) {
    SimValues->IcellIDX->data[i0] = Icells->data[i0];
  }

  emxFree_real_T(&Icells);

  /* SimValues.WeightMat       = EE_mat+II_mat+EI_mat+IE_mat; */
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  CUDACHECK(hipFree(gpu_recordIntervals));
  CUDACHECK(hipFree(gpu_saveINT));

#undef CUDACHECK

}

/* End of code generation (CAdLIFfunction_STDP.cu) */
